#include "hip/hip_runtime.h"
// This file is part of OpenMVG, an Open Multiple View Geometry C++ library.

// Copyright (c) 2016 Kareem Omar (kareem.omar@uah.edu - https://github.com/komrad36)

// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

// Fastest GPU implementation of a brute-force
// Hamming-weight matrix for 512-bit binary descriptors.
//
// Yes, that means the DIFFERENCE in popcounts is used
// for thresholding, NOT the ratio. This is the CORRECT
// approach for binary descriptors.
//
// This laboriously crafted kernel is EXTREMELY fast.
// 63 BILLION comparisons per second on a stock GTX1080,
// enough to match nearly 46,000 descriptors per frame at 30 fps (!)
//
// A key insight responsible for much of the performance of
// this insanely fast CUDA kernel is due to
// Christopher Parker (https://github.com/csp256), to whom
// I am extremely grateful.
//
// CUDA CC 3.0 or higher is required.


#include "CudaBruteForceMatcher.h"

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
cudaBruteForceMatcher_kernel
(
  const hipTextureObject_t tex_q,
  const int num_q,
  const uint64_t* __restrict__ g_training,
  const int num_t,
  int* const __restrict__ g_match,
  const int threshold
 )
 {
  uint64_t train = *(g_training += threadIdx.x & 7);
  g_training += 8;
  uint64_t q[8];
  for (int i = 0, offset = ((threadIdx.x & 24) << 3) + (threadIdx.x & 7) + (blockIdx.x << 11) + (threadIdx.y << 8); i < 8; ++i, offset += 8)
  {
    const uint2 buf = tex1Dfetch<uint2>(tex_q, offset);
    asm("mov.b64 %0, {%1,%2};" : "=l"(q[i]) : "r"(buf.x), "r"(buf.y)); // some assembly required
  }
  int best_i, best_v = 100000, second_v = 200000;
#pragma unroll 6
  for (int t = 0; t < num_t; ++t, g_training += 8)
  {
    uint32_t dist[4];
    for (int i = 0; i < 4; ++i) dist[i] = __byte_perm(__popcll(q[i] ^ train), __popcll(q[i + 4] ^ train), 0x5410);
    for (int i = 0; i < 4; ++i) dist[i] += __shfl_xor(dist[i], 1);
    train = *g_training;
    if (threadIdx.x & 1) dist[0] = dist[1];
    if (threadIdx.x & 1) dist[2] = dist[3];
    dist[0] += __shfl_xor(dist[0], 2);
    dist[2] += __shfl_xor(dist[2], 2);
    if (threadIdx.x & 2) dist[0] = dist[2];
    dist[0] = __byte_perm(dist[0] + __shfl_xor(dist[0], 4), 0, threadIdx.x & 4 ? 0x5432 : 0x5410);
    second_v = min(dist[0], second_v);
    if (dist[0] < best_v)
    {
      second_v = best_v;
      best_i = t;
      best_v = dist[0];
    }
  }
  const int idx = (blockIdx.x << 8) + (threadIdx.y << 5) + threadIdx.x;
  if (idx < num_q) g_match[idx] = second_v - best_v > threshold ? best_i : -1;
}

void cudaBruteForceMatcher
(
  const void* const __restrict d_t,
  const int num_t,
  const hipTextureObject_t tex_q,
  const int num_q,
  int* const __restrict d_m,
  const int threshold,
  const hipStream_t stream
)
{
  cudaBruteForceMatcher_kernel<<<((num_q - 1) >> 8) + 1, { 32, 8 }, 0, stream>>>(tex_q, num_q, reinterpret_cast<const uint64_t*>(d_t), num_t, d_m, threshold);
}