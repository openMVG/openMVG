#include "hip/hip_runtime.h"
//********************************************************//
// CUDA SIFT extractor by Marten Bjorkman aka Celebrandil //
//********************************************************//  

#include "cudautils.h"
#include "cudaSiftD.h"
#include "cudaSift.h"

///////////////////////////////////////////////////////////////////////////////
// Kernel configuration
///////////////////////////////////////////////////////////////////////////////

__constant__ int d_MaxNumPoints;
__device__ unsigned int d_PointCounter[8*2+1];
__constant__ float d_ScaleDownKernel[5]; 
__constant__ float d_LowPassKernel[2*LOWPASS_R+1]; 
__constant__ float d_LaplaceKernel[8*12*16]; 

///////////////////////////////////////////////////////////////////////////////
// Lowpass filter and subsample image
///////////////////////////////////////////////////////////////////////////////
__global__ void ScaleDownDenseShift(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
#define BW (SCALEDOWN_W+4)
#define BH (SCALEDOWN_H+4)
#define W2 (SCALEDOWN_W/2)
#define H2 (SCALEDOWN_H/2)
  __shared__ float brows[BH*BW];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int xp = blockIdx.x*SCALEDOWN_W + tx;
  const int yp = blockIdx.y*SCALEDOWN_H + ty;
  const float k0 = d_ScaleDownKernel[0];
  const float k1 = d_ScaleDownKernel[1];
  const float k2 = d_ScaleDownKernel[2];
  const int xl = min(width-1,  max(0, xp-2));
  const int yl = min(height-1, max(0, yp-2));
  if (xp<(width+4) && yp<(height+4)) {
    float v = d_Data[yl*pitch + xl];
    brows[BW*ty + tx]  = k0*(v + ShiftDown(v, 4)) + k1*(ShiftDown(v, 1) + ShiftDown(v, 3)) + k2*ShiftDown(v, 2);
  }
  __syncthreads();
  const int xs = blockIdx.x*W2 + tx;
  const int ys = blockIdx.y*H2 + ty;
  if (tx<W2 && ty<H2 && xs<(width/2) && ys<(height/2)) {
    float *ptr = &brows[BW*(ty*2) + (tx*2)];
    d_Result[ys*newpitch + xs] = k0*(ptr[0] + ptr[4*BW]) + k1*(ptr[1*BW] + ptr[3*BW]) + k2*ptr[2*BW];
  } 
}

__global__ void ScaleDownDense(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
#define BW (SCALEDOWN_W+4)
#define BH (SCALEDOWN_H+4)
#define W2 (SCALEDOWN_W/2)
#define H2 (SCALEDOWN_H/2)
  __shared__ float irows[BH*BW]; 
  __shared__ float brows[BH*W2];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int xp = blockIdx.x*SCALEDOWN_W + tx;
  const int yp = blockIdx.y*SCALEDOWN_H + ty;
  const int xl = min(width-1,  max(0, xp-2));
  const int yl = min(height-1, max(0, yp-2));
  const float k0 = d_ScaleDownKernel[0];
  const float k1 = d_ScaleDownKernel[1];
  const float k2 = d_ScaleDownKernel[2];
  if (xp<(width+4) && yp<(height+4))
    irows[BW*ty + tx] = d_Data[yl*pitch + xl];
  __syncthreads();
  if (yp<(height+4) && tx<W2) {
    float *ptr = &irows[BW*ty + 2*tx];
    brows[W2*ty + tx] = k0*(ptr[0] + ptr[4]) + k1*(ptr[1] + ptr[3]) + k2*ptr[2];
  }
  __syncthreads();
  const int xs = blockIdx.x*W2 + tx;
  const int ys = blockIdx.y*H2 + ty;
  if (tx<W2 && ty<H2 && xs<(width/2) && ys<(height/2)) {
    float *ptr = &brows[W2*(ty*2) + tx];
    d_Result[ys*newpitch + xs] = k0*(ptr[0] + ptr[4*W2]) + k1*(ptr[1*W2] + ptr[3*W2]) + k2*ptr[2*W2];
  } 
}

__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
  __shared__ float inrow[SCALEDOWN_W+4]; 
  __shared__ float brow[5*(SCALEDOWN_W/2)];
  __shared__ int yRead[SCALEDOWN_H+4];
  __shared__ int yWrite[SCALEDOWN_H+4];
  #define dx2 (SCALEDOWN_W/2)
  const int tx = threadIdx.x;
  const int tx0 = tx + 0*dx2;
  const int tx1 = tx + 1*dx2;
  const int tx2 = tx + 2*dx2;
  const int tx3 = tx + 3*dx2;
  const int tx4 = tx + 4*dx2;
  const int xStart = blockIdx.x*SCALEDOWN_W;
  const int yStart = blockIdx.y*SCALEDOWN_H;
  const int xWrite = xStart/2 + tx;
  float k0 = d_ScaleDownKernel[0];
  float k1 = d_ScaleDownKernel[1];
  float k2 = d_ScaleDownKernel[2];
  if (tx<SCALEDOWN_H+4) {
    int y = yStart + tx - 2; 
    y = (y<0 ? 0 : y);
    y = (y>=height ? height-1 : y);
    yRead[tx] = y*pitch;
    yWrite[tx] = (yStart + tx - 4)/2 * newpitch;
  }
  __syncthreads();
  int xRead = xStart + tx - 2;
  xRead = (xRead<0 ? 0 : xRead);
  xRead = (xRead>=width ? width-1 : xRead);

  int maxtx = min(dx2, width/2 - xStart/2);
  for (int dy=0;dy<SCALEDOWN_H+4;dy+=5) {
    {
      inrow[tx] = d_Data[yRead[dy+0] + xRead];
      __syncthreads();
      if (tx<maxtx) {
	brow[tx4] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
	if (dy>=4 && !(dy&1))
	  d_Result[yWrite[dy+0] + xWrite] = k2*brow[tx2] + k0*(brow[tx0]+brow[tx4]) + k1*(brow[tx1]+brow[tx3]);
      }
      __syncthreads();
    }
    if (dy<(SCALEDOWN_H+3)) {
      inrow[tx] = d_Data[yRead[dy+1] + xRead];
      __syncthreads();
      if (tx<maxtx) {
	brow[tx0] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
	if (dy>=3 && (dy&1))
	  d_Result[yWrite[dy+1] + xWrite] = k2*brow[tx3] + k0*(brow[tx1]+brow[tx0]) + k1*(brow[tx2]+brow[tx4]);
      }
      __syncthreads();
    }
    if (dy<(SCALEDOWN_H+2)) {
      inrow[tx] = d_Data[yRead[dy+2] + xRead];
      __syncthreads();
      if (tx<maxtx) {
	brow[tx1] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
	if (dy>=2 && !(dy&1))
	  d_Result[yWrite[dy+2] + xWrite] = k2*brow[tx4] + k0*(brow[tx2]+brow[tx1]) + k1*(brow[tx3]+brow[tx0]);
      }
      __syncthreads();
    }
    if (dy<(SCALEDOWN_H+1)) {
      inrow[tx] = d_Data[yRead[dy+3] + xRead];
      __syncthreads();
      if (tx<maxtx) {
	brow[tx2] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
	if (dy>=1 && (dy&1))
	  d_Result[yWrite[dy+3] + xWrite] = k2*brow[tx0] + k0*(brow[tx3]+brow[tx2]) + k1*(brow[tx4]+brow[tx1]);
      }
      __syncthreads();
    }
    if (dy<SCALEDOWN_H) {
      inrow[tx] = d_Data[yRead[dy+4] + xRead];
      __syncthreads();
      if (tx<dx2 && xWrite<width/2) {
	brow[tx3] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
	if (!(dy&1))
	  d_Result[yWrite[dy+4] + xWrite] = k2*brow[tx1] + k0*(brow[tx4]+brow[tx3]) + k1*(brow[tx0]+brow[tx2]);
      }
      __syncthreads();
    }
  }
}

__global__ void ScaleUp(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
  #undef BW
  #undef BH
  #define BW (SCALEUP_W/2 + 2)
  #define BH (SCALEUP_H/2 + 2)
  __shared__ float buffer[BW*BH];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  if (tx<BW && ty<BH) {
    int x = min(max(blockIdx.x*(SCALEUP_W/2) + tx - 1, 0), width-1);
    int y = min(max(blockIdx.y*(SCALEUP_H/2) + ty - 1, 0), height-1);
    buffer[ty*BW + tx] = d_Data[y*pitch + x];
  }
  __syncthreads();
  int x = blockIdx.x*SCALEUP_W + tx;
  int y = blockIdx.y*SCALEUP_H + ty;
  if (x<2*width && y<2*height) {
    int bx = (tx + 1)/2;
    int by = (ty + 1)/2;
    int bp = by*BW + bx;
    float wx = 0.25f + (tx&1)*0.50f;
    float wy = 0.25f + (ty&1)*0.50f;
    d_Result[y*newpitch + x] = wy*(wx*buffer[bp] + (1.0f-wx)*buffer[bp+1]) +
      (1.0f-wy)*(wx*buffer[bp+BW] + (1.0f-wx)*buffer[bp+BW+1]);
  }
}

__global__ void ExtractSiftDescriptors(hipTextureObject_t texObj, SiftPoint *d_sift, int fstPts, float subsampling)
{
  __shared__ float gauss[16];
  __shared__ float buffer[128];
  __shared__ float sums[4];

  const int tx = threadIdx.x; // 0 -> 16
  const int ty = threadIdx.y; // 0 -> 8
  const int idx = ty*16 + tx;
  const int bx = blockIdx.x + fstPts;  // 0 -> numPts
  if (ty==0)
    gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);
  buffer[idx] = 0.0f;
  __syncthreads();

  // Compute angles and gradients
  float theta = 2.0f*3.1415f/360.0f*d_sift[bx].orientation;
  float sina = sinf(theta);           // cosa -sina
  float cosa = cosf(theta);           // sina  cosa
  float scale = 12.0f/16.0f*d_sift[bx].scale;
  float ssina = scale*sina; 
  float scosa = scale*cosa;

  for (int y=ty;y<16;y+=8) {
    float xpos = d_sift[bx].xpos + (tx-7.5f)*scosa - (y-7.5f)*ssina + 0.5f;
    float ypos = d_sift[bx].ypos + (tx-7.5f)*ssina + (y-7.5f)*scosa + 0.5f;
    float dx = tex2D<float>(texObj, xpos+cosa, ypos+sina) - 
      tex2D<float>(texObj, xpos-cosa, ypos-sina);
    float dy = tex2D<float>(texObj, xpos-sina, ypos+cosa) - 
      tex2D<float>(texObj, xpos+sina, ypos-cosa);
    float grad = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
    float angf = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;
    
    int hori = (tx + 2)/4 - 1;      // Convert from (tx,y,angle) to bins      
    float horf = (tx - 1.5f)/4.0f - hori;
    float ihorf = 1.0f - horf;           
    int veri = (y + 2)/4 - 1;
    float verf = (y - 1.5f)/4.0f - veri;
    float iverf = 1.0f - verf;
    int angi = angf;
    int angp = (angi<7 ? angi+1 : 0);
    angf -= angi;
    float iangf = 1.0f - angf;
    
    int hist = 8*(4*veri + hori);   // Each gradient measure is interpolated 
    int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
    int p2 = angp + hist;
    if (tx>=2) { 
      float grad1 = ihorf*grad;
      if (y>=2) {   // Upper left
        float grad2 = iverf*grad1;
	atomicAdd(buffer + p1, iangf*grad2);
	atomicAdd(buffer + p2,  angf*grad2);
      }
      if (y<=13) {  // Lower left
        float grad2 = verf*grad1;
	atomicAdd(buffer + p1+32, iangf*grad2); 
	atomicAdd(buffer + p2+32,  angf*grad2);
      }
    }
    if (tx<=13) { 
      float grad1 = horf*grad;
      if (y>=2) {    // Upper right
        float grad2 = iverf*grad1;
	atomicAdd(buffer + p1+8, iangf*grad2);
	atomicAdd(buffer + p2+8,  angf*grad2);
      }
      if (y<=13) {   // Lower right
        float grad2 = verf*grad1;
	atomicAdd(buffer + p1+40, iangf*grad2);
	atomicAdd(buffer + p2+40,  angf*grad2);
      }
    }
  }
  __syncthreads();

  // Normalize twice and suppress peaks first time
  float sum = buffer[idx]*buffer[idx];
  for (int i=16;i>0;i/=2)
    sum += ShiftDown(sum, i);
  if ((idx&31)==0)
    sums[idx/32] = sum;
  __syncthreads();
  float tsum1 = sums[0] + sums[1] + sums[2] + sums[3]; 
  tsum1 = min(buffer[idx] * rsqrtf(tsum1), 0.2f);
  
  sum = tsum1*tsum1; 
  for (int i=16;i>0;i/=2)
    sum += ShiftDown(sum, i);
  if ((idx&31)==0)
    sums[idx/32] = sum;
  __syncthreads();

  float tsum2 = sums[0] + sums[1] + sums[2] + sums[3];
  float *desc = d_sift[bx].data;
  desc[idx] = tsum1 * rsqrtf(tsum2);
  if (idx==0) {
    d_sift[bx].xpos *= subsampling;
    d_sift[bx].ypos *= subsampling;
    d_sift[bx].scale *= subsampling;
  }
}
 

__global__ void ExtractSiftDescriptorsCONST(hipTextureObject_t texObj, SiftPoint *d_sift, float subsampling, int octave)
{
  __shared__ float gauss[16];
  __shared__ float buffer[128];
  __shared__ float sums[4];

  const int tx = threadIdx.x; // 0 -> 16
  const int ty = threadIdx.y; // 0 -> 8
  const int idx = ty*16 + tx;
  if (ty==0)
    gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);

  int fstPts = min(d_PointCounter[2*octave-1], d_MaxNumPoints);
  int totPts = min(d_PointCounter[2*octave+1], d_MaxNumPoints);
  //if (tx==0 && ty==0)
  //  printf("%d %d %d %d\n", octave, fstPts, min(d_PointCounter[2*octave], d_MaxNumPoints), totPts); 
  for (int bx = blockIdx.x + fstPts; bx < totPts; bx += gridDim.x) {
    
    buffer[idx] = 0.0f;
    __syncthreads();

    // Compute angles and gradients
    float theta = 2.0f*3.1415f/360.0f*d_sift[bx].orientation;
    float sina = sinf(theta);           // cosa -sina
    float cosa = cosf(theta);           // sina  cosa
    float scale = 12.0f/16.0f*d_sift[bx].scale;
    float ssina = scale*sina; 
    float scosa = scale*cosa;
    
    for (int y=ty;y<16;y+=8) {
      float xpos = d_sift[bx].xpos + (tx-7.5f)*scosa - (y-7.5f)*ssina + 0.5f; 
      float ypos = d_sift[bx].ypos + (tx-7.5f)*ssina + (y-7.5f)*scosa + 0.5f;
      float dx = tex2D<float>(texObj, xpos+cosa, ypos+sina) - 
	tex2D<float>(texObj, xpos-cosa, ypos-sina);
      float dy = tex2D<float>(texObj, xpos-sina, ypos+cosa) - 
	tex2D<float>(texObj, xpos+sina, ypos-cosa);
      float grad = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
      float angf = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;
      
      int hori = (tx + 2)/4 - 1;      // Convert from (tx,y,angle) to bins      
      float horf = (tx - 1.5f)/4.0f - hori;
      float ihorf = 1.0f - horf;           
      int veri = (y + 2)/4 - 1;
      float verf = (y - 1.5f)/4.0f - veri;
      float iverf = 1.0f - verf;
      int angi = angf;
      int angp = (angi<7 ? angi+1 : 0);
      angf -= angi;
      float iangf = 1.0f - angf;
      
      int hist = 8*(4*veri + hori);   // Each gradient measure is interpolated 
      int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
      int p2 = angp + hist;
      if (tx>=2) { 
	float grad1 = ihorf*grad;
	if (y>=2) {   // Upper left
	  float grad2 = iverf*grad1;
	  atomicAdd(buffer + p1, iangf*grad2);
	  atomicAdd(buffer + p2,  angf*grad2);
	}
	if (y<=13) {  // Lower left
	  float grad2 = verf*grad1;
	  atomicAdd(buffer + p1+32, iangf*grad2); 
	  atomicAdd(buffer + p2+32,  angf*grad2);
	}
      }
      if (tx<=13) { 
	float grad1 = horf*grad;
	if (y>=2) {    // Upper right
	  float grad2 = iverf*grad1;
	  atomicAdd(buffer + p1+8, iangf*grad2);
	  atomicAdd(buffer + p2+8,  angf*grad2);
	}
	if (y<=13) {   // Lower right
	  float grad2 = verf*grad1;
	  atomicAdd(buffer + p1+40, iangf*grad2);
	  atomicAdd(buffer + p2+40,  angf*grad2);
	}
      }
    }
    __syncthreads();
    
    // Normalize twice and suppress peaks first time
    float sum = buffer[idx]*buffer[idx];
    for (int i=16;i>0;i/=2)
      sum += ShiftDown(sum, i);
    if ((idx&31)==0)
      sums[idx/32] = sum;
    __syncthreads();
    float tsum1 = sums[0] + sums[1] + sums[2] + sums[3]; 
    tsum1 = min(buffer[idx] * rsqrtf(tsum1), 0.2f);
     
    sum = tsum1*tsum1; 
    for (int i=16;i>0;i/=2)
      sum += ShiftDown(sum, i);
    if ((idx&31)==0)
      sums[idx/32] = sum;
    __syncthreads();
    
    float tsum2 = sums[0] + sums[1] + sums[2] + sums[3];
    float *desc = d_sift[bx].data;
    desc[idx] = tsum1 * rsqrtf(tsum2);
    if (idx==0) {
      d_sift[bx].xpos *= subsampling;
      d_sift[bx].ypos *= subsampling;
      d_sift[bx].scale *= subsampling;
    }
    __syncthreads();
  }
}
 

__global__ void ExtractSiftDescriptorsOld(hipTextureObject_t texObj, SiftPoint *d_sift, int fstPts, float subsampling)
{
  __shared__ float gauss[16];
  __shared__ float buffer[128];
  __shared__ float sums[128];

  const int tx = threadIdx.x; // 0 -> 16
  const int ty = threadIdx.y; // 0 -> 8
  const int idx = ty*16 + tx;
  const int bx = blockIdx.x + fstPts;  // 0 -> numPts
  if (ty==0)
    gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);
  buffer[idx] = 0.0f;
  __syncthreads();

  // Compute angles and gradients
  float theta = 2.0f*3.1415f/360.0f*d_sift[bx].orientation;
  float sina = sinf(theta);           // cosa -sina
  float cosa = cosf(theta);           // sina  cosa
  float scale = 12.0f/16.0f*d_sift[bx].scale;
  float ssina = scale*sina; 
  float scosa = scale*cosa;

  for (int y=ty;y<16;y+=8) {
    float xpos = d_sift[bx].xpos + (tx-7.5f)*scosa - (y-7.5f)*ssina + 0.5f;
    float ypos = d_sift[bx].ypos + (tx-7.5f)*ssina + (y-7.5f)*scosa + 0.5f;
    float dx = tex2D<float>(texObj, xpos+cosa, ypos+sina) - 
      tex2D<float>(texObj, xpos-cosa, ypos-sina);
    float dy = tex2D<float>(texObj, xpos-sina, ypos+cosa) - 
      tex2D<float>(texObj, xpos+sina, ypos-cosa);
    float grad = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
    float angf = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;
    
    int hori = (tx + 2)/4 - 1;      // Convert from (tx,y,angle) to bins      
    float horf = (tx - 1.5f)/4.0f - hori;  
    float ihorf = 1.0f - horf;           
    int veri = (y + 2)/4 - 1;
    float verf = (y - 1.5f)/4.0f - veri;
    float iverf = 1.0f - verf;
    int angi = angf;
    int angp = (angi<7 ? angi+1 : 0);
    angf -= angi;
    float iangf = 1.0f - angf;
    
    int hist = 8*(4*veri + hori);   // Each gradient measure is interpolated 
    int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
    int p2 = angp + hist;
    if (tx>=2) { 
      float grad1 = ihorf*grad;
      if (y>=2) {   // Upper left
        float grad2 = iverf*grad1;
	atomicAdd(buffer + p1, iangf*grad2);
	atomicAdd(buffer + p2,  angf*grad2);
      }
      if (y<=13) {  // Lower left
        float grad2 = verf*grad1;
	atomicAdd(buffer + p1+32, iangf*grad2); 
	atomicAdd(buffer + p2+32,  angf*grad2);
      }
    }
    if (tx<=13) { 
      float grad1 = horf*grad;
      if (y>=2) {    // Upper right
        float grad2 = iverf*grad1;
	atomicAdd(buffer + p1+8, iangf*grad2);
	atomicAdd(buffer + p2+8,  angf*grad2);
      }
      if (y<=13) {   // Lower right
        float grad2 = verf*grad1;
	atomicAdd(buffer + p1+40, iangf*grad2);
	atomicAdd(buffer + p2+40,  angf*grad2);
      }
    }
  }
  __syncthreads();

  // Normalize twice and suppress peaks first time
  if (idx<64)
    sums[idx] = buffer[idx]*buffer[idx] + buffer[idx+64]*buffer[idx+64];
  __syncthreads();      
  if (idx<32) sums[idx] = sums[idx] + sums[idx+32];
  __syncthreads();      
  if (idx<16) sums[idx] = sums[idx] + sums[idx+16];
  __syncthreads();      
  if (idx<8)  sums[idx] = sums[idx] + sums[idx+8];
  __syncthreads();      
  if (idx<4)  sums[idx] = sums[idx] + sums[idx+4];
  __syncthreads();      
  float tsum1 = sums[0] + sums[1] + sums[2] + sums[3]; 
  buffer[idx] = buffer[idx] * rsqrtf(tsum1);

  if (buffer[idx]>0.2f)
    buffer[idx] = 0.2f;
  __syncthreads();
  if (idx<64)
    sums[idx] = buffer[idx]*buffer[idx] + buffer[idx+64]*buffer[idx+64];
  __syncthreads();      
  if (idx<32) sums[idx] = sums[idx] + sums[idx+32];
  __syncthreads();      
  if (idx<16) sums[idx] = sums[idx] + sums[idx+16];
  __syncthreads();      
  if (idx<8)  sums[idx] = sums[idx] + sums[idx+8];
  __syncthreads();      
  if (idx<4)  sums[idx] = sums[idx] + sums[idx+4];
  __syncthreads();      
  float tsum2 = sums[0] + sums[1] + sums[2] + sums[3]; 

  float *desc = d_sift[bx].data;
  desc[idx] = buffer[idx] * rsqrtf(tsum2);
  if (idx==0) {
    d_sift[bx].xpos *= subsampling;
    d_sift[bx].ypos *= subsampling;
    d_sift[bx].scale *= subsampling;
  }
}


__device__ void ExtractSiftDescriptor(hipTextureObject_t texObj, SiftPoint *d_sift, float subsampling, int octave, int bx)
{
  __shared__ float gauss[16];
  __shared__ float buffer[128];
  __shared__ float sums[4];

  const int idx = threadIdx.x;
  const int tx = idx & 15; // 0 -> 16
  const int ty = idx / 16; // 0 -> 8
  if (ty==0)
    gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);
  buffer[idx] = 0.0f;
  __syncthreads();

  // Compute angles and gradients
  float theta = 2.0f*3.1415f/360.0f*d_sift[bx].orientation;
  float sina = sinf(theta);           // cosa -sina
  float cosa = cosf(theta);           // sina  cosa
  float scale = 12.0f/16.0f*d_sift[bx].scale;
  float ssina = scale*sina; 
  float scosa = scale*cosa;
  
  for (int y=ty;y<16;y+=8) {
    float xpos = d_sift[bx].xpos + (tx-7.5f)*scosa - (y-7.5f)*ssina + 0.5f;
    float ypos = d_sift[bx].ypos + (tx-7.5f)*ssina + (y-7.5f)*scosa + 0.5f;
    float dx = tex2D<float>(texObj, xpos+cosa, ypos+sina) - 
      tex2D<float>(texObj, xpos-cosa, ypos-sina);
    float dy = tex2D<float>(texObj, xpos-sina, ypos+cosa) - 
      tex2D<float>(texObj, xpos+sina, ypos-cosa);
    float grad = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
    float angf = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;
    
    int hori = (tx + 2)/4 - 1;      // Convert from (tx,y,angle) to bins      
    float horf = (tx - 1.5f)/4.0f - hori;
    float ihorf = 1.0f - horf;           
    int veri = (y + 2)/4 - 1;
    float verf = (y - 1.5f)/4.0f - veri;
    float iverf = 1.0f - verf;
    int angi = angf;
    int angp = (angi<7 ? angi+1 : 0);
    angf -= angi;
    float iangf = 1.0f - angf;
    
    int hist = 8*(4*veri + hori);   // Each gradient measure is interpolated 
    int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
    int p2 = angp + hist;
    if (tx>=2) { 
      float grad1 = ihorf*grad;
      if (y>=2) {   // Upper left
	float grad2 = iverf*grad1;
	atomicAdd(buffer + p1, iangf*grad2);
	atomicAdd(buffer + p2,  angf*grad2);
      }
      if (y<=13) {  // Lower left
	float grad2 = verf*grad1;
	atomicAdd(buffer + p1+32, iangf*grad2); 
	atomicAdd(buffer + p2+32,  angf*grad2);
      }
    }
    if (tx<=13) { 
      float grad1 = horf*grad;
      if (y>=2) {    // Upper right
	float grad2 = iverf*grad1;
	atomicAdd(buffer + p1+8, iangf*grad2);
	atomicAdd(buffer + p2+8,  angf*grad2);
      }
      if (y<=13) {   // Lower right
	float grad2 = verf*grad1;
	atomicAdd(buffer + p1+40, iangf*grad2);
	atomicAdd(buffer + p2+40,  angf*grad2);
      }
    }
  }
  __syncthreads();
    
  // Normalize twice and suppress peaks first time
  float sum = buffer[idx]*buffer[idx];
  for (int i=16;i>0;i/=2)
    sum += ShiftDown(sum, i);
  if ((idx&31)==0)
    sums[idx/32] = sum;
  __syncthreads();
  float tsum1 = sums[0] + sums[1] + sums[2] + sums[3]; 
  tsum1 = min(buffer[idx] * rsqrtf(tsum1), 0.2f);
  
  sum = tsum1*tsum1; 
  for (int i=16;i>0;i/=2)
    sum += ShiftDown(sum, i);
  if ((idx&31)==0)
    sums[idx/32] = sum;
  __syncthreads();
  
  float tsum2 = sums[0] + sums[1] + sums[2] + sums[3];
  float *desc = d_sift[bx].data;
  desc[idx] = tsum1 * rsqrtf(tsum2);
  if (idx==0) {
    d_sift[bx].xpos *= subsampling;
    d_sift[bx].ypos *= subsampling;
    d_sift[bx].scale *= subsampling;
  }
  __syncthreads();
}


__global__ void RescalePositions(SiftPoint *d_sift, int numPts, float scale)
{
  int num = blockIdx.x*blockDim.x + threadIdx.x;
  if (num<numPts) {
    d_sift[num].xpos *= scale;
    d_sift[num].ypos *= scale;
    d_sift[num].scale *= scale;
  }
}


__global__ void ComputeOrientations(hipTextureObject_t texObj, SiftPoint *d_Sift, int fstPts)
{
  __shared__ float hist[64];
  __shared__ float gauss[11];
  const int tx = threadIdx.x;
  const int bx = blockIdx.x + fstPts;
  float i2sigma2 = -1.0f/(4.5f*d_Sift[bx].scale*d_Sift[bx].scale);
  if (tx<11) 
    gauss[tx] = exp(i2sigma2*(tx-5)*(tx-5));
  if (tx<64)
    hist[tx] = 0.0f;
  __syncthreads();
  float xp = d_Sift[bx].xpos - 4.5f;
  float yp = d_Sift[bx].ypos - 4.5f;
  int yd = tx/11;
  int xd = tx - yd*11;
  float xf = xp + xd;
  float yf = yp + yd;
  if (yd<11) {
    float dx = tex2D<float>(texObj, xf+1.0, yf) - tex2D<float>(texObj, xf-1.0, yf); 
    float dy = tex2D<float>(texObj, xf, yf+1.0) - tex2D<float>(texObj, xf, yf-1.0); 
    int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
    if (bin>31)
      bin = 0;
    float grad = sqrtf(dx*dx + dy*dy);
    atomicAdd(&hist[bin], grad*gauss[xd]*gauss[yd]);
  }
  __syncthreads();
  int x1m = (tx>=1 ? tx-1 : tx+31);
  int x1p = (tx<=30 ? tx+1 : tx-31);
  if (tx<32) {
    int x2m = (tx>=2 ? tx-2 : tx+30);
    int x2p = (tx<=29 ? tx+2 : tx-30);
    hist[tx+32] = 6.0f*hist[tx] + 4.0f*(hist[x1m] + hist[x1p]) + (hist[x2m] + hist[x2p]);
  }
  __syncthreads();
  if (tx<32) {
    float v = hist[32+tx];
    hist[tx] = (v>hist[32+x1m] && v>=hist[32+x1p] ? v : 0.0f);
  }
  __syncthreads();
  if (tx==0) {
    float maxval1 = 0.0;
    float maxval2 = 0.0;
    int i1 = -1;
    int i2 = -1;
    for (int i=0;i<32;i++) {
      float v = hist[i];
      if (v>maxval1) {
	maxval2 = maxval1;
	maxval1 = v;
	i2 = i1;
	i1 = i;
      } else if (v>maxval2) {
	maxval2 = v;
	i2 = i;
      }
    }
    float val1 = hist[32+((i1+1)&31)];
    float val2 = hist[32+((i1+31)&31)];
    float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
    d_Sift[bx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);
    if (maxval2>0.8f*maxval1) {
      float val1 = hist[32+((i2+1)&31)];
      float val2 = hist[32+((i2+31)&31)];
      float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
      unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
      if (idx<d_MaxNumPoints) {
	d_Sift[idx].xpos = d_Sift[bx].xpos;
	d_Sift[idx].ypos = d_Sift[bx].ypos;
	d_Sift[idx].scale = d_Sift[bx].scale;
	d_Sift[idx].sharpness = d_Sift[bx].sharpness;
	d_Sift[idx].edgeness = d_Sift[bx].edgeness;
	d_Sift[idx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);;
	d_Sift[idx].subsampling = d_Sift[bx].subsampling;
      }
    } 
  }
} 

// With constant number of blocks
__global__ void ComputeOrientationsCONST(hipTextureObject_t texObj, SiftPoint *d_Sift, int octave)
{
  __shared__ float hist[64];
  __shared__ float gauss[11];
  const int tx = threadIdx.x;
  
  int fstPts = min(d_PointCounter[2*octave-1], d_MaxNumPoints);
  int totPts = min(d_PointCounter[2*octave+0], d_MaxNumPoints);  
  for (int bx = blockIdx.x + fstPts; bx < totPts; bx += gridDim.x) {
 
    float i2sigma2 = -1.0f/(4.5f*d_Sift[bx].scale*d_Sift[bx].scale);
    if (tx<11) 
      gauss[tx] = exp(i2sigma2*(tx-5)*(tx-5));
    if (tx<64)
      hist[tx] = 0.0f;
    __syncthreads();
    float xp = d_Sift[bx].xpos - 4.5f;
    float yp = d_Sift[bx].ypos - 4.5f;
    int yd = tx/11;
    int xd = tx - yd*11;
    float xf = xp + xd;
    float yf = yp + yd;
    if (yd<11) {
      float dx = tex2D<float>(texObj, xf+1.0, yf) - tex2D<float>(texObj, xf-1.0, yf); 
      float dy = tex2D<float>(texObj, xf, yf+1.0) - tex2D<float>(texObj, xf, yf-1.0); 
      int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
      if (bin>31)
	bin = 0;
      float grad = sqrtf(dx*dx + dy*dy);
      atomicAdd(&hist[bin], grad*gauss[xd]*gauss[yd]);
    }
    __syncthreads();
    int x1m = (tx>=1 ? tx-1 : tx+31);
    int x1p = (tx<=30 ? tx+1 : tx-31);
    if (tx<32) {
      int x2m = (tx>=2 ? tx-2 : tx+30);
      int x2p = (tx<=29 ? tx+2 : tx-30);
      hist[tx+32] = 6.0f*hist[tx] + 4.0f*(hist[x1m] + hist[x1p]) + (hist[x2m] + hist[x2p]);
    }
    __syncthreads();
    if (tx<32) {
      float v = hist[32+tx];
      hist[tx] = (v>hist[32+x1m] && v>=hist[32+x1p] ? v : 0.0f);
    }
    __syncthreads();
    if (tx==0) {
      float maxval1 = 0.0;
      float maxval2 = 0.0;
      int i1 = -1;
      int i2 = -1;
      for (int i=0;i<32;i++) {
	float v = hist[i];
	if (v>maxval1) {
	  maxval2 = maxval1;
	  maxval1 = v;
	  i2 = i1;
	  i1 = i;
	} else if (v>maxval2) {
	  maxval2 = v;
	  i2 = i;
	}
      }
      float val1 = hist[32+((i1+1)&31)];
      float val2 = hist[32+((i1+31)&31)];
      float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
      d_Sift[bx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);
      atomicMax(&d_PointCounter[2*octave+1], d_PointCounter[2*octave+0]); 
      if (maxval2>0.8f*maxval1) {
	float val1 = hist[32+((i2+1)&31)];
	float val2 = hist[32+((i2+31)&31)];
	float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
	unsigned int idx = atomicInc(&d_PointCounter[2*octave+1], 0x7fffffff);
	if (idx<d_MaxNumPoints) {
	  d_Sift[idx].xpos = d_Sift[bx].xpos;
	  d_Sift[idx].ypos = d_Sift[bx].ypos;
	  d_Sift[idx].scale = d_Sift[bx].scale;
	  d_Sift[idx].sharpness = d_Sift[bx].sharpness;
	  d_Sift[idx].edgeness = d_Sift[bx].edgeness;
	  d_Sift[idx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);;
	  d_Sift[idx].subsampling = d_Sift[bx].subsampling;
	}
      }
    }
    __syncthreads();
  }
} 

// With constant number of blocks
__global__ void OrientAndExtractCONST(hipTextureObject_t texObj, SiftPoint *d_Sift, float subsampling, int octave)
{
  __shared__ float hist[64];
  __shared__ float gauss[11];
  __shared__ unsigned int idx; //%%%%
  const int tx = threadIdx.x;
  
  int fstPts = min(d_PointCounter[2*octave-1], d_MaxNumPoints);
  int totPts = min(d_PointCounter[2*octave+0], d_MaxNumPoints);  
  for (int bx = blockIdx.x + fstPts; bx < totPts; bx += gridDim.x) {
 
    float i2sigma2 = -1.0f/(4.5f*d_Sift[bx].scale*d_Sift[bx].scale);
    if (tx<11) 
      gauss[tx] = exp(i2sigma2*(tx-5)*(tx-5));
    if (tx<64)
      hist[tx] = 0.0f;
    __syncthreads();
    float xp = d_Sift[bx].xpos - 4.5f;
    float yp = d_Sift[bx].ypos - 4.5f;
    int yd = tx/11;
    int xd = tx - yd*11;
    float xf = xp + xd;
    float yf = yp + yd;
    if (yd<11) {
      float dx = tex2D<float>(texObj, xf+1.0, yf) - tex2D<float>(texObj, xf-1.0, yf); 
      float dy = tex2D<float>(texObj, xf, yf+1.0) - tex2D<float>(texObj, xf, yf-1.0); 
      int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
      if (bin>31)
	bin = 0;
      float grad = sqrtf(dx*dx + dy*dy);
      atomicAdd(&hist[bin], grad*gauss[xd]*gauss[yd]);
    }
    __syncthreads();
    int x1m = (tx>=1 ? tx-1 : tx+31);
    int x1p = (tx<=30 ? tx+1 : tx-31);
    if (tx<32) {
      int x2m = (tx>=2 ? tx-2 : tx+30);
      int x2p = (tx<=29 ? tx+2 : tx-30);
      hist[tx+32] = 6.0f*hist[tx] + 4.0f*(hist[x1m] + hist[x1p]) + (hist[x2m] + hist[x2p]);
    }
    __syncthreads();
    if (tx<32) {
      float v = hist[32+tx];
      hist[tx] = (v>hist[32+x1m] && v>=hist[32+x1p] ? v : 0.0f);
    }
    __syncthreads();
    if (tx==0) {
      float maxval1 = 0.0;
      float maxval2 = 0.0;
      int i1 = -1;
      int i2 = -1;
      for (int i=0;i<32;i++) {
	float v = hist[i];
	if (v>maxval1) {
	  maxval2 = maxval1;
	  maxval1 = v;
	  i2 = i1;
	  i1 = i;
	} else if (v>maxval2) {
	  maxval2 = v;
	  i2 = i;
	}
      }
      float val1 = hist[32+((i1+1)&31)];
      float val2 = hist[32+((i1+31)&31)];
      float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
      d_Sift[bx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);
      idx = 0xffffffff; //%%%%
      atomicMax(&d_PointCounter[2*octave+1], d_PointCounter[2*octave+0]); 
      if (maxval2>0.8f*maxval1) {
	float val1 = hist[32+((i2+1)&31)];
	float val2 = hist[32+((i2+31)&31)];
	float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
	idx = atomicInc(&d_PointCounter[2*octave+1], 0x7fffffff); //%%%%
	if (idx<d_MaxNumPoints) {
	  d_Sift[idx].xpos = d_Sift[bx].xpos;
	  d_Sift[idx].ypos = d_Sift[bx].ypos;
	  d_Sift[idx].scale = d_Sift[bx].scale;
	  d_Sift[idx].sharpness = d_Sift[bx].sharpness;
	  d_Sift[idx].edgeness = d_Sift[bx].edgeness;
	  d_Sift[idx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);;
	  d_Sift[idx].subsampling = d_Sift[bx].subsampling;
	}
      }
    }
    __syncthreads();
    ExtractSiftDescriptor(texObj, d_Sift, subsampling, octave, bx); //%%%%
    if (idx<d_MaxNumPoints) //%%%%
      ExtractSiftDescriptor(texObj, d_Sift, subsampling, octave, idx); //%%%%
  }
} 


///////////////////////////////////////////////////////////////////////////////
// Subtract two images (multi-scale version)
///////////////////////////////////////////////////////////////////////////////
  
__global__ void FindPointsMultiTest(float *d_Data0, SiftPoint *d_Sift, int width, int pitch, int height, float subsampling, float lowestScale, float thresh, float factor, float edgeLimit, int octave)
{
  #define MEMWID (MINMAX_W + 2)
  __shared__ unsigned int cnt;
  __shared__ unsigned short points[3*MEMWID];

  if (blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0 && threadIdx.y==0) {
    atomicMax(&d_PointCounter[2*octave+0], d_PointCounter[2*octave-1]); 
    atomicMax(&d_PointCounter[2*octave+1], d_PointCounter[2*octave-1]);
  }
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  if (tx==0 && ty==0)
    cnt = 0; 
  __syncthreads();

  int ypos = MINMAX_H*blockIdx.y + ty;
  if (ypos>=height)
    return;
  int block = blockIdx.x/NUM_SCALES; 
  int scale = blockIdx.x - NUM_SCALES*block;
  int minx = block*MINMAX_W;
  int maxx = min(minx + MINMAX_W, width);
  int xpos = minx + tx;
  int size = pitch*height;
  int ptr = size*scale + max(min(xpos-1, width-1), 0);

  float maxv = fabs(d_Data0[ptr + ypos*pitch + 1*size]);
  maxv = fmaxf(maxv, ShiftDown(maxv, 16, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 8, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 4, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 2, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 1, MINMAX_W));
    
  if (Shuffle(maxv, 0)>thresh) {
    int yptr1 = ptr + ypos*pitch;
    int yptr0 = ptr + max(0,ypos-1)*pitch;
    int yptr2 = ptr + min(height-1,ypos+1)*pitch;
    float d20 = d_Data0[yptr0 + 1*size];
    float d21 = d_Data0[yptr1 + 1*size];
    float d22 = d_Data0[yptr2 + 1*size];
    float d31 = d_Data0[yptr1 + 2*size];
    float d11 = d_Data0[yptr1];
    
    float d10 = d_Data0[yptr0];
    float d12 = d_Data0[yptr2];
    float ymin1 = fminf(fminf(d10, d11), d12);
    float ymax1 = fmaxf(fmaxf(d10, d11), d12);
    float d30 = d_Data0[yptr0 + 2*size];
    float d32 = d_Data0[yptr2 + 2*size]; 
    float ymin3 = fminf(fminf(d30, d31), d32);
    float ymax3 = fmaxf(fmaxf(d30, d31), d32);
    float ymin2 = fminf(fminf(ymin1, fminf(fminf(d20, d22), d21)), ymin3);
    float ymax2 = fmaxf(fmaxf(ymax1, fmaxf(fmaxf(d20, d22), d21)), ymax3);
    
    float nmin2 = fminf(ShiftUp(ymin2, 1), ShiftDown(ymin2, 1));
    float nmax2 = fmaxf(ShiftUp(ymax2, 1), ShiftDown(ymax2, 1));
    if (tx>0 && tx<MINMAX_W+1 && xpos<=maxx) {
      if (d21<-thresh) {
	float minv = fminf(fminf(nmin2, ymin1), ymin3);
	minv = fminf(fminf(minv, d20), d22);
	if (d21<minv) { 
	  int pos = atomicInc(&cnt, MEMWID-1);
	  points[3*pos+0] = xpos - 1;
	  points[3*pos+1] = ypos;
	  points[3*pos+2] = scale;
	}
      } 
      if (d21>thresh) {
	float maxv = fmaxf(fmaxf(nmax2, ymax1), ymax3);
	maxv = fmaxf(fmaxf(maxv, d20), d22);
	if (d21>maxv) { 
	  int pos = atomicInc(&cnt, MEMWID-1);
	  points[3*pos+0] = xpos - 1;
	  points[3*pos+1] = ypos;
	  points[3*pos+2] = scale;
	}
      }
    }
  }
  __syncthreads();
  if (ty==0 && tx<cnt) {
    int xpos = points[3*tx+0];
    int ypos = points[3*tx+1];
    int scale = points[3*tx+2];
    int ptr = xpos + (ypos + (scale+1)*height)*pitch;
    float val = d_Data0[ptr];
    float *data1 = &d_Data0[ptr];
    float dxx = 2.0f*val - data1[-1] - data1[1];
    float dyy = 2.0f*val - data1[-pitch] - data1[pitch];
    float dxy = 0.25f*(data1[+pitch+1] + data1[-pitch-1] - data1[-pitch+1] - data1[+pitch-1]);
    float tra = dxx + dyy;
    float det = dxx*dyy - dxy*dxy;
    if (tra*tra<edgeLimit*det) {
      float edge = __fdividef(tra*tra, det);
      float dx = 0.5f*(data1[1] - data1[-1]);
      float dy = 0.5f*(data1[pitch] - data1[-pitch]); 
      float *data0 = d_Data0 + ptr - height*pitch;
      float *data2 = d_Data0 + ptr + height*pitch;
      float ds = 0.5f*(data0[0] - data2[0]); 
      float dss = 2.0f*val - data2[0] - data0[0];
      float dxs = 0.25f*(data2[1] + data0[-1] - data0[1] - data2[-1]);
      float dys = 0.25f*(data2[pitch] + data0[-pitch] - data2[-pitch] - data0[pitch]);
      float idxx = dyy*dss - dys*dys;
      float idxy = dys*dxs - dxy*dss;   
      float idxs = dxy*dys - dyy*dxs;
      float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
      float idyy = dxx*dss - dxs*dxs;
      float idys = dxy*dxs - dxx*dys;
      float idss = dxx*dyy - dxy*dxy;
      float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
      float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
      float pds = idet*(idxs*dx + idys*dy + idss*ds);
      if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
	pdx = __fdividef(dx, dxx);
	pdy = __fdividef(dy, dyy);
	pds = __fdividef(ds, dss);
      }
      float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
      int maxPts = d_MaxNumPoints;
      float sc = powf(2.0f, (float)scale/NUM_SCALES) * exp2f(pds*factor);
      if (sc>=lowestScale) {
	unsigned int idx = atomicInc(&d_PointCounter[2*octave+0], 0x7fffffff);
	idx = (idx>=maxPts ? maxPts-1 : idx);
	d_Sift[idx].xpos = xpos + pdx;
	d_Sift[idx].ypos = ypos + pdy;
	d_Sift[idx].scale = sc;
	d_Sift[idx].sharpness = val + dval;
	d_Sift[idx].edgeness = edge;
	d_Sift[idx].subsampling = subsampling;
      }
    }
  }
}

__global__ void FindPointsMulti(float *d_Data0, SiftPoint *d_Sift, int width, int pitch, int height, float subsampling, float lowestScale, float thresh, float factor, float edgeLimit, int octave)
{
  #define MEMWID (MINMAX_W + 2)
  __shared__ unsigned int cnt;
  __shared__ unsigned short points[3*MEMWID];

  if (blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0) {
    atomicMax(&d_PointCounter[2*octave+0], d_PointCounter[2*octave-1]);
    atomicMax(&d_PointCounter[2*octave+1], d_PointCounter[2*octave-1]);
  }
  int tx = threadIdx.x;
  int block = blockIdx.x/NUM_SCALES; 
  int scale = blockIdx.x - NUM_SCALES*block;
  int minx = block*MINMAX_W;
  int maxx = min(minx + MINMAX_W, width);
  int xpos = minx + tx;
  int size = pitch*height;
  int ptr = size*scale + max(min(xpos-1, width-1), 0);

  int yloops = min(height - MINMAX_H*blockIdx.y, MINMAX_H);
  float maxv = 0.0f;
  for (int y=0;y<yloops;y++) {
    int ypos = MINMAX_H*blockIdx.y + y;
    int yptr1 = ptr + ypos*pitch;
    float val = d_Data0[yptr1 + 1*size];
    maxv = fmaxf(maxv, fabs(val));
  }
  maxv = fmaxf(maxv, ShiftDown(maxv, 16, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 8, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 4, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 2, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 1, MINMAX_W));
  if (Shuffle(maxv, 0)<=thresh)
    return;
  
  if (tx==0)
    cnt = 0; 
  __syncthreads();

  for (int y=0;y<yloops;y++) {

    int ypos = MINMAX_H*blockIdx.y + y;
    int yptr1 = ptr + ypos*pitch;
    int yptr0 = ptr + max(0,ypos-1)*pitch;
    int yptr2 = ptr + min(height-1,ypos+1)*pitch;
    float d20 = d_Data0[yptr0 + 1*size];
    float d21 = d_Data0[yptr1 + 1*size];
    float d22 = d_Data0[yptr2 + 1*size];
    float d31 = d_Data0[yptr1 + 2*size];
    float d11 = d_Data0[yptr1];
    
    float d10 = d_Data0[yptr0];
    float d12 = d_Data0[yptr2];
    float ymin1 = fminf(fminf(d10, d11), d12);
    float ymax1 = fmaxf(fmaxf(d10, d11), d12);
    float d30 = d_Data0[yptr0 + 2*size];
    float d32 = d_Data0[yptr2 + 2*size]; 
    float ymin3 = fminf(fminf(d30, d31), d32);
    float ymax3 = fmaxf(fmaxf(d30, d31), d32);
    float ymin2 = fminf(fminf(ymin1, fminf(fminf(d20, d22), d21)), ymin3);
    float ymax2 = fmaxf(fmaxf(ymax1, fmaxf(fmaxf(d20, d22), d21)), ymax3);
    
    float nmin2 = fminf(ShiftUp(ymin2, 1), ShiftDown(ymin2, 1));
    float nmax2 = fmaxf(ShiftUp(ymax2, 1), ShiftDown(ymax2, 1));
    if (tx>0 && tx<MINMAX_W+1 && xpos<=maxx) {
      if (d21<-thresh) {
	float minv = fminf(fminf(nmin2, ymin1), ymin3);
	minv = fminf(fminf(minv, d20), d22);
	if (d21<minv) { 
	  int pos = atomicInc(&cnt, MEMWID-1);
	  points[3*pos+0] = xpos - 1;
	  points[3*pos+1] = ypos;
	  points[3*pos+2] = scale;
	}
      } 
      if (d21>thresh) {
	float maxv = fmaxf(fmaxf(nmax2, ymax1), ymax3);
	maxv = fmaxf(fmaxf(maxv, d20), d22);
	if (d21>maxv) { 
	  int pos = atomicInc(&cnt, MEMWID-1);
	  points[3*pos+0] = xpos - 1;
	  points[3*pos+1] = ypos;
	  points[3*pos+2] = scale;
	}
      }
    }
  }
  if (tx<cnt) {
    int xpos = points[3*tx+0];
    int ypos = points[3*tx+1];
    int scale = points[3*tx+2];
    int ptr = xpos + (ypos + (scale+1)*height)*pitch;
    float val = d_Data0[ptr];
    float *data1 = &d_Data0[ptr];
    float dxx = 2.0f*val - data1[-1] - data1[1];
    float dyy = 2.0f*val - data1[-pitch] - data1[pitch];
    float dxy = 0.25f*(data1[+pitch+1] + data1[-pitch-1] - data1[-pitch+1] - data1[+pitch-1]);
    float tra = dxx + dyy;
    float det = dxx*dyy - dxy*dxy;
    if (tra*tra<edgeLimit*det) {
      float edge = __fdividef(tra*tra, det);
      float dx = 0.5f*(data1[1] - data1[-1]);
      float dy = 0.5f*(data1[pitch] - data1[-pitch]); 
      float *data0 = d_Data0 + ptr - height*pitch;
      float *data2 = d_Data0 + ptr + height*pitch;
      float ds = 0.5f*(data0[0] - data2[0]); 
      float dss = 2.0f*val - data2[0] - data0[0];
      float dxs = 0.25f*(data2[1] + data0[-1] - data0[1] - data2[-1]);
      float dys = 0.25f*(data2[pitch] + data0[-pitch] - data2[-pitch] - data0[pitch]);
      float idxx = dyy*dss - dys*dys;
      float idxy = dys*dxs - dxy*dss;   
      float idxs = dxy*dys - dyy*dxs;
      float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
      float idyy = dxx*dss - dxs*dxs;
      float idys = dxy*dxs - dxx*dys;
      float idss = dxx*dyy - dxy*dxy;
      float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
      float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
      float pds = idet*(idxs*dx + idys*dy + idss*ds);
      if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
	pdx = __fdividef(dx, dxx);
	pdy = __fdividef(dy, dyy);
	pds = __fdividef(ds, dss);
      }
      float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
      int maxPts = d_MaxNumPoints;
      float sc = powf(2.0f, (float)scale/NUM_SCALES) * exp2f(pds*factor);
      if (sc>=lowestScale) {
	atomicMax(&d_PointCounter[2*octave+0], d_PointCounter[2*octave-1]); 
	unsigned int idx = atomicInc(&d_PointCounter[2*octave+0], 0x7fffffff);
	idx = (idx>=maxPts ? maxPts-1 : idx);
	d_Sift[idx].xpos = xpos + pdx;
	d_Sift[idx].ypos = ypos + pdy;
	d_Sift[idx].scale = sc;
	d_Sift[idx].sharpness = val + dval;
	d_Sift[idx].edgeness = edge;
	d_Sift[idx].subsampling = subsampling;
      }
    }
  }
}


__global__ void FindPointsMultiOld(float *d_Data0, SiftPoint *d_Sift, int width, int pitch, int height, float subsampling, float lowestScale, float thresh, float factor, float edgeLimit, int octave)
{
  #define MEMWID (MINMAX_W + 2)
  __shared__ float ymin1[MEMWID], ymin2[MEMWID], ymin3[MEMWID];
  __shared__ float ymax1[MEMWID], ymax2[MEMWID], ymax3[MEMWID];
  __shared__ unsigned int cnt;
  __shared__ unsigned short points[3*MEMWID];

  if (blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0) {
    atomicMax(&d_PointCounter[2*octave+0], d_PointCounter[2*octave-1]); 
    atomicMax(&d_PointCounter[2*octave+1], d_PointCounter[2*octave-1]);
  }
  int tx = threadIdx.x;
  int block = blockIdx.x/NUM_SCALES; 
  int scale = blockIdx.x - NUM_SCALES*block;
  int minx = block*MINMAX_W;
  int maxx = min(minx + MINMAX_W, width);
  int xpos = minx + tx;
  int size = pitch*height;
  int ptr = size*scale + max(min(xpos-1, width-1), 0);

  int yloops = min(height - MINMAX_H*blockIdx.y, MINMAX_H);
  float maxv = 0.0f;
  for (int y=0;y<yloops;y++) {
    int ypos = MINMAX_H*blockIdx.y + y;
    int yptr1 = ptr + ypos*pitch;
    float val = d_Data0[yptr1 + 1*size];
    maxv = fmaxf(maxv, fabs(val));
  }
  maxv = fmaxf(maxv, ShiftDown(maxv, 16, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 8, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 4, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 2, MINMAX_W));
  maxv = fmaxf(maxv, ShiftDown(maxv, 1, MINMAX_W));
  if (Shuffle(maxv, 0)<=thresh)
    return;
  
  if (tx==0)
    cnt = 0; 
  __syncthreads();

  for (int y=0;y<yloops;y++) {

    int ypos = MINMAX_H*blockIdx.y + y;
    int yptr1 = ptr + ypos*pitch;
    int yptr0 = ptr + max(0,ypos-1)*pitch;
    int yptr2 = ptr + min(height-1,ypos+1)*pitch;
    float d20 = d_Data0[yptr0 + 1*size];
    float d21 = d_Data0[yptr1 + 1*size];
    float d22 = d_Data0[yptr2 + 1*size];
    float d31 = d_Data0[yptr1 + 2*size];
    float d11 = d_Data0[yptr1];

    float d10 = d_Data0[yptr0];
    float d12 = d_Data0[yptr2];
    ymin1[tx] = fminf(fminf(d10, d11), d12);
    ymax1[tx] = fmaxf(fmaxf(d10, d11), d12);
    float d30 = d_Data0[yptr0 + 2*size];
    float d32 = d_Data0[yptr2 + 2*size]; 
    ymin3[tx] = fminf(fminf(d30, d31), d32);
    ymax3[tx] = fmaxf(fmaxf(d30, d31), d32);
    ymin2[tx] = fminf(fminf(ymin1[tx], fminf(fminf(d20, d22), d21)), ymin3[tx]);
    ymax2[tx] = fmaxf(fmaxf(ymax1[tx], fmaxf(fmaxf(d20, d22), d21)), ymax3[tx]);
    
    __syncthreads(); 

    if (tx>0 && tx<MINMAX_W+1 && xpos<=maxx) {
      if (d21<-thresh) {
	float minv = fminf(fminf(fminf(ymin2[tx-1], ymin2[tx+1]), ymin1[tx]), ymin3[tx]);
	minv = fminf(fminf(minv, d20), d22);
	if (d21<minv) { 
	  int pos = atomicInc(&cnt, MEMWID-1);
	  points[3*pos+0] = xpos - 1;
	  points[3*pos+1] = ypos;
	  points[3*pos+2] = scale;
	}
      } 
      if (d21>thresh) {
	float maxv = fmaxf(fmaxf(fmaxf(ymax2[tx-1], ymax2[tx+1]), ymax1[tx]), ymax3[tx]);
	maxv = fmaxf(fmaxf(maxv, d20), d22);
	if (d21>maxv) { 
	  int pos = atomicInc(&cnt, MEMWID-1);
	  points[3*pos+0] = xpos - 1;
	  points[3*pos+1] = ypos;
	  points[3*pos+2] = scale;
	}
      }
    }
    __syncthreads();
  }
  if (tx<cnt) {
    int xpos = points[3*tx+0];
    int ypos = points[3*tx+1];
    int scale = points[3*tx+2];
    int ptr = xpos + (ypos + (scale+1)*height)*pitch;
    float val = d_Data0[ptr];
    float *data1 = &d_Data0[ptr];
    float dxx = 2.0f*val - data1[-1] - data1[1];
    float dyy = 2.0f*val - data1[-pitch] - data1[pitch];
    float dxy = 0.25f*(data1[+pitch+1] + data1[-pitch-1] - data1[-pitch+1] - data1[+pitch-1]);
    float tra = dxx + dyy;
    float det = dxx*dyy - dxy*dxy;
    if (tra*tra<edgeLimit*det) {
      float edge = __fdividef(tra*tra, det);
      float dx = 0.5f*(data1[1] - data1[-1]);
      float dy = 0.5f*(data1[pitch] - data1[-pitch]); 
      float *data0 = d_Data0 + ptr - height*pitch;
      float *data2 = d_Data0 + ptr + height*pitch;
      float ds = 0.5f*(data0[0] - data2[0]); 
      float dss = 2.0f*val - data2[0] - data0[0];
      float dxs = 0.25f*(data2[1] + data0[-1] - data0[1] - data2[-1]);
      float dys = 0.25f*(data2[pitch] + data0[-pitch] - data2[-pitch] - data0[pitch]);
      float idxx = dyy*dss - dys*dys;
      float idxy = dys*dxs - dxy*dss;   
      float idxs = dxy*dys - dyy*dxs;
      float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
      float idyy = dxx*dss - dxs*dxs;
      float idys = dxy*dxs - dxx*dys;
      float idss = dxx*dyy - dxy*dxy;
      float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
      float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
      float pds = idet*(idxs*dx + idys*dy + idss*ds);
      if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
	pdx = __fdividef(dx, dxx);
	pdy = __fdividef(dy, dyy);
	pds = __fdividef(ds, dss);
      }
      float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
      int maxPts = d_MaxNumPoints;
      float sc = powf(2.0f, (float)scale/NUM_SCALES) * exp2f(pds*factor);
      if (sc>=lowestScale) {
	unsigned int idx = atomicInc(&d_PointCounter[2*octave+0], 0x7fffffff);
	idx = (idx>=maxPts ? maxPts-1 : idx);
	d_Sift[idx].xpos = xpos + pdx;
	d_Sift[idx].ypos = ypos + pdy;
	d_Sift[idx].scale = sc;
	d_Sift[idx].sharpness = val + dval;
	d_Sift[idx].edgeness = edge;
	d_Sift[idx].subsampling = subsampling;
      }
    }
  }
}


__global__ void LaplaceMultiTex(hipTextureObject_t texObj, float *d_Result, int width, int pitch, int height, int octave)
{
  __shared__ float data1[(LAPLACE_W + 2*LAPLACE_R)*LAPLACE_S];
  __shared__ float data2[LAPLACE_W*LAPLACE_S];
  const int tx = threadIdx.x;
  const int xp = blockIdx.x*LAPLACE_W + tx;
  const int yp = blockIdx.y;
  const int scale = threadIdx.y;
  float *kernel = d_LaplaceKernel + octave*12*16 + scale*16;
  float *sdata1 = data1 + (LAPLACE_W + 2*LAPLACE_R)*scale; 
  float x = xp-3.5;
  float y = yp+0.5;
  sdata1[tx] = kernel[4]*tex2D<float>(texObj, x, y) + 
    kernel[3]*(tex2D<float>(texObj, x, y-1.0) + tex2D<float>(texObj, x, y+1.0)) + 
    kernel[2]*(tex2D<float>(texObj, x, y-2.0) + tex2D<float>(texObj, x, y+2.0)) + 
    kernel[1]*(tex2D<float>(texObj, x, y-3.0) + tex2D<float>(texObj, x, y+3.0)) + 
    kernel[0]*(tex2D<float>(texObj, x, y-4.0) + tex2D<float>(texObj, x, y+4.0));
  __syncthreads();
  float *sdata2 = data2 + LAPLACE_W*scale; 
  if (tx<LAPLACE_W) {
    sdata2[tx] = kernel[4]*sdata1[tx+4] + 
      kernel[3]*(sdata1[tx+3] + sdata1[tx+5]) + 
      kernel[2]*(sdata1[tx+2] + sdata1[tx+6]) + 
      kernel[1]*(sdata1[tx+1] + sdata1[tx+7]) + 
      kernel[0]*(sdata1[tx+0] + sdata1[tx+8]);
  }
  __syncthreads(); 
  if (tx<LAPLACE_W && scale<LAPLACE_S-1 && xp<width) 
    d_Result[scale*height*pitch + yp*pitch + xp] = sdata2[tx] - sdata2[tx+LAPLACE_W];
}


__global__ void LaplaceMultiMem(float *d_Image, float *d_Result, int width, int pitch, int height, int octave)
{
  __shared__ float buff[(LAPLACE_W + 2*LAPLACE_R)*LAPLACE_S];
  const int tx = threadIdx.x;
  const int xp = blockIdx.x*LAPLACE_W + tx;
  const int yp = blockIdx.y;
  float *data = d_Image + max(min(xp - 4, width-1), 0);
  float temp[9], kern[LAPLACE_S][LAPLACE_R+1];
  if (xp<(width + 2*LAPLACE_R)) {
    for (int i=0;i<4;i++)
      temp[i] = data[max(0, min(yp+i-4, height-1))*pitch];
    for (int i=4;i<8+1;i++)
      temp[i] = data[min(yp+i-4, height-1)*pitch];
    for (int scale=0;scale<LAPLACE_S;scale++) {
      float *buf = buff + (LAPLACE_W + 2*LAPLACE_R)*scale; 
      float *kernel = d_LaplaceKernel + octave*12*16 + scale*16; 
      for (int i=0;i<LAPLACE_R+1;i++)
	kern[scale][i] = kernel[i];
      buf[tx] = kern[scale][4]*temp[4] +
	kern[scale][3]*(temp[3] + temp[5]) + kern[scale][2]*(temp[2] + temp[6]) + 
	kern[scale][1]*(temp[1] + temp[7]) + kern[scale][0]*(temp[0] + temp[8]);
    }
  }
  __syncthreads();
  if (tx<LAPLACE_W && xp<width) {
    float oldRes = kern[0][4]*buff[tx+4] + 
      kern[0][3]*(buff[tx+3] + buff[tx+5]) + kern[0][2]*(buff[tx+2] + buff[tx+6]) + 
      kern[0][1]*(buff[tx+1] + buff[tx+7]) + kern[0][0]*(buff[tx+0] + buff[tx+8]);
    for (int scale=1;scale<LAPLACE_S;scale++) {
      float *buf = buff + (LAPLACE_W + 2*LAPLACE_R)*scale; 
      float res = kern[scale][4]*buf[tx+4] + 
	kern[scale][3]*(buf[tx+3] + buf[tx+5]) + kern[scale][2]*(buf[tx+2] + buf[tx+6]) + 
	kern[scale][1]*(buf[tx+1] + buf[tx+7]) + kern[scale][0]*(buf[tx+0] + buf[tx+8]);
      d_Result[(scale-1)*height*pitch + yp*pitch + xp] = res - oldRes;
      oldRes = res;
    }
  }
}

__global__ void LaplaceMultiMemTest(float *d_Image, float *d_Result, int width, int pitch, int height, int octave)
{
  __shared__ float data1[(LAPLACE_W + 2*LAPLACE_R)*LAPLACE_S];
  __shared__ float data2[LAPLACE_W*LAPLACE_S];
  const int tx = threadIdx.x;
  const int xp = blockIdx.x*LAPLACE_W + tx;
  const int yp = LAPLACE_H*blockIdx.y;
  const int scale = threadIdx.y;
  float *kernel = d_LaplaceKernel + octave*12*16 + scale*16; 
  float *sdata1 = data1 + (LAPLACE_W + 2*LAPLACE_R)*scale; 
  float *data = d_Image + max(min(xp - 4, width-1), 0);
  int h = height-1;
  float temp[8+LAPLACE_H], kern[LAPLACE_R+1];
  for (int i=0;i<4;i++)
    temp[i] = data[max(0, min(yp+i-4, h))*pitch];
  for (int i=4;i<8+LAPLACE_H;i++)
    temp[i] = data[min(yp+i-4, h)*pitch];
  for (int i=0;i<LAPLACE_R+1;i++)
    kern[i] = kernel[i];
  for (int j=0;j<LAPLACE_H;j++) {
    sdata1[tx] = kern[4]*temp[4+j] +
      kern[3]*(temp[3+j] + temp[5+j]) + kern[2]*(temp[2+j] + temp[6+j]) + 
      kern[1]*(temp[1+j] + temp[7+j]) + kern[0]*(temp[0+j] + temp[8+j]);
    __syncthreads();
    float *sdata2 = data2 + LAPLACE_W*scale; 
    if (tx<LAPLACE_W) {
      sdata2[tx] = kern[4]*sdata1[tx+4] + 
	kern[3]*(sdata1[tx+3] + sdata1[tx+5]) + kern[2]*(sdata1[tx+2] + sdata1[tx+6]) + 
	kern[1]*(sdata1[tx+1] + sdata1[tx+7]) + kern[0]*(sdata1[tx+0] + sdata1[tx+8]);
    }
    __syncthreads(); 
    if (tx<LAPLACE_W && scale<LAPLACE_S-1 && xp<width && (yp+j)<height) 
      d_Result[scale*height*pitch + (yp+j)*pitch + xp] = sdata2[tx] - sdata2[tx+LAPLACE_W];
  }
}

__global__ void LaplaceMultiMemOld(float *d_Image, float *d_Result, int width, int pitch, int height, int octave)
{
  __shared__ float data1[(LAPLACE_W + 2*LAPLACE_R)*LAPLACE_S];
  __shared__ float data2[LAPLACE_W*LAPLACE_S];
  const int tx = threadIdx.x;
  const int xp = blockIdx.x*LAPLACE_W + tx;
  const int yp = blockIdx.y;
  const int scale = threadIdx.y;
  float *kernel = d_LaplaceKernel + octave*12*16 + scale*16; 
  float *sdata1 = data1 + (LAPLACE_W + 2*LAPLACE_R)*scale; 
  float *data = d_Image + max(min(xp - 4, width-1), 0);
  int h = height-1;
  sdata1[tx] = kernel[4]*data[min(yp, h)*pitch] +
    kernel[3]*(data[max(0, min(yp-1, h))*pitch] + data[min(yp+1, h)*pitch]) + 
    kernel[2]*(data[max(0, min(yp-2, h))*pitch] + data[min(yp+2, h)*pitch]) + 
    kernel[1]*(data[max(0, min(yp-3, h))*pitch] + data[min(yp+3, h)*pitch]) + 
    kernel[0]*(data[max(0, min(yp-4, h))*pitch] + data[min(yp+4, h)*pitch]);
  __syncthreads();
  float *sdata2 = data2 + LAPLACE_W*scale; 
  if (tx<LAPLACE_W) {
    sdata2[tx] = kernel[4]*sdata1[tx+4] + 
      kernel[3]*(sdata1[tx+3] + sdata1[tx+5]) + kernel[2]*(sdata1[tx+2] + sdata1[tx+6]) + 
      kernel[1]*(sdata1[tx+1] + sdata1[tx+7]) + kernel[0]*(sdata1[tx+0] + sdata1[tx+8]);
  }
  __syncthreads(); 
  if (tx<LAPLACE_W && scale<LAPLACE_S-1 && xp<width) 
    d_Result[scale*height*pitch + yp*pitch + xp] = sdata2[tx] - sdata2[tx+LAPLACE_W];
}

__global__ void LowPass(float *d_Image, float *d_Result, int width, int pitch, int height)
{
  __shared__ float buffer[(LOWPASS_W + 2*LOWPASS_R)*LOWPASS_H];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int xp = blockIdx.x*LOWPASS_W + tx;
  const int yp = blockIdx.y*LOWPASS_H + ty;
  float *kernel = d_LowPassKernel;
  float *data = d_Image + max(min(xp - 4, width-1), 0);
  float *buff = buffer + ty*(LOWPASS_W + 2*LOWPASS_R);
  int h = height-1;
  if (yp<height) 
    buff[tx] = kernel[4]*data[min(yp, h)*pitch] +
      kernel[3]*(data[max(0, min(yp-1, h))*pitch] + data[min(yp+1, h)*pitch]) + 
      kernel[2]*(data[max(0, min(yp-2, h))*pitch] + data[min(yp+2, h)*pitch]) + 
      kernel[1]*(data[max(0, min(yp-3, h))*pitch] + data[min(yp+3, h)*pitch]) + 
      kernel[0]*(data[max(0, min(yp-4, h))*pitch] + data[min(yp+4, h)*pitch]);
  __syncthreads();
  if (tx<LOWPASS_W && xp<width && yp<height)
    d_Result[yp*pitch + xp] = kernel[4]*buff[tx+4] + 
      kernel[3]*(buff[tx+3] + buff[tx+5]) + kernel[2]*(buff[tx+2] + buff[tx+6]) + 
      kernel[1]*(buff[tx+1] + buff[tx+7]) + kernel[0]*(buff[tx+0] + buff[tx+8]);
}

__global__ void LowPassBlockOld(float *d_Image, float *d_Result, int width, int pitch, int height)
{
  __shared__ float xrows[16][32];          
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int xp = blockIdx.x*LOWPASS_W + tx;
  const int yp = blockIdx.y*LOWPASS_H + ty;
  const int N = 16;
  float *k = d_LowPassKernel;
  int xl = max(min(xp - 4, width-1), 0);
  for (int l=-8;l<=LOWPASS_H;l+=4) {
    if (l<LOWPASS_H) {
      int yl = max(min(yp + l + 4, height-1), 0);
      float val = d_Image[yl*pitch + xl];
      xrows[(l + 8 + ty)%N][tx] = k[4]*ShiftDown(val, 4) +
	k[3]*(ShiftDown(val, 5) + ShiftDown(val, 3)) +
	k[2]*(ShiftDown(val, 6) + ShiftDown(val, 2)) +
	k[1]*(ShiftDown(val, 7) + ShiftDown(val, 1)) +
	k[0]*(ShiftDown(val, 8) + val);
    }
    if (l>=4) {
      int ys = yp + l - 4;
      if (xp<width && ys<height && tx<LOWPASS_W)
	d_Result[ys*pitch + xp] = k[4]*xrows[(l + 0 + ty)%N][tx] +
	     k[3]*(xrows[(l - 1 + ty)%N][tx] + xrows[(l + 1 + ty)%N][tx]) +
	     k[2]*(xrows[(l - 2 + ty)%N][tx] + xrows[(l + 2 + ty)%N][tx]) +
	     k[1]*(xrows[(l - 3 + ty)%N][tx] + xrows[(l + 3 + ty)%N][tx]) +
	     k[0]*(xrows[(l - 4 + ty)%N][tx] + xrows[(l + 4 + ty)%N][tx]);
    }
    if (l>=0)
      __syncthreads();
  }
}

__global__ void LowPassBlock(float *d_Image, float *d_Result, int width, int pitch, int height)
{
  __shared__ float xrows[16][32];          
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int xp = blockIdx.x*LOWPASS_W + tx;
  const int yp = blockIdx.y*LOWPASS_H + ty;
  const int N = 16;
  float *k = d_LowPassKernel;
  int xl = max(min(xp - 4, width-1), 0);
#pragma unroll
  for (int l=-8;l<4;l+=4) {
    int ly = l + ty;
    int yl = max(min(yp + l + 4, height-1), 0);
    float val = d_Image[yl*pitch + xl];
    val = k[4]*ShiftDown(val, 4) +
      k[3]*(ShiftDown(val, 5) + ShiftDown(val, 3)) +
      k[2]*(ShiftDown(val, 6) + ShiftDown(val, 2)) +
      k[1]*(ShiftDown(val, 7) + ShiftDown(val, 1)) +
      k[0]*(ShiftDown(val, 8) + val);
    xrows[ly + 8][tx] = val;
  }
  __syncthreads();
#pragma unroll
  for (int l=4;l<LOWPASS_H;l+=4) {
    int ly = l + ty;
    int yl = min(yp + l + 4, height-1);
    float val = d_Image[yl*pitch + xl];
    val = k[4]*ShiftDown(val, 4) +
      k[3]*(ShiftDown(val, 5) + ShiftDown(val, 3)) +
      k[2]*(ShiftDown(val, 6) + ShiftDown(val, 2)) +
      k[1]*(ShiftDown(val, 7) + ShiftDown(val, 1)) +
      k[0]*(ShiftDown(val, 8) + val);
    xrows[(ly + 8)%N][tx] = val;
    int ys = yp + l - 4;
    if (xp<width && ys<height && tx<LOWPASS_W)
      d_Result[ys*pitch + xp] = k[4]*xrows[(ly + 0)%N][tx] +
		       k[3]*(xrows[(ly - 1)%N][tx] + xrows[(ly + 1)%N][tx]) +
		       k[2]*(xrows[(ly - 2)%N][tx] + xrows[(ly + 2)%N][tx]) +
		       k[1]*(xrows[(ly - 3)%N][tx] + xrows[(ly + 3)%N][tx]) +
		       k[0]*(xrows[(ly - 4)%N][tx] + xrows[(ly + 4)%N][tx]);
    __syncthreads();
  }
  int ly = LOWPASS_H + ty;
  int ys = yp + LOWPASS_H - 4;
  if (xp<width && ys<height && tx<LOWPASS_W)
    d_Result[ys*pitch + xp] = k[4]*xrows[(ly + 0)%N][tx] +
		     k[3]*(xrows[(ly - 1)%N][tx] + xrows[(ly + 1)%N][tx]) +
		     k[2]*(xrows[(ly - 2)%N][tx] + xrows[(ly + 2)%N][tx]) +
		     k[1]*(xrows[(ly - 3)%N][tx] + xrows[(ly + 3)%N][tx]) +
		     k[0]*(xrows[(ly - 4)%N][tx] + xrows[(ly + 4)%N][tx]);
}

