#include "hip/hip_runtime.h"
//********************************************************//
// CUDA SIFT extractor by Mårten Björkman aka Celebrandil //
//********************************************************//  

#include <cstdio>
#include <cstring>
#include <cmath>
#include <iostream>
#include <algorithm>
#include "cudautils.h"



#include "cudaImage.h"
#include "cudaSift.h"
#include "cudaSiftD.h"
#include "cudaSiftH.h"

#include "cudaSiftD.cu"

void InitCuda(int devNum)
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  if (!nDevices) {
    std::cerr << "No CUDA devices available" << std::endl;
    return;
  }
  devNum = std::min(nDevices-1, devNum);
  deviceInit(devNum);  
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, devNum);
  printf("Device Number: %d\n", devNum);
  printf("  Device name: %s\n", prop.name);
  printf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate/1000);
  printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
  printf("  Peak Memory Bandwidth (GB/s): %.1f\n\n",
	 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
}

void ExtractSift(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, bool scaleUp) 
{
  TimerGPU timer(0);
  int totPts = 0;
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_PointCounter), &totPts, sizeof(int)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_MaxNumPoints), &siftData.maxPts, sizeof(int)));

  const int nd = NUM_SCALES + 3;
  int w = img.width*(scaleUp ? 2 : 1);
  int h = img.height*(scaleUp ? 2 : 1);
  int p = iAlignUp(w, 128);
  int width = w, height = h;
  int size = h*p;                 // image sizes
  int sizeTmp = nd*h*p;           // laplace buffer sizes
  for (int i=0;i<numOctaves;i++) {
    w /= 2;
    h /= 2;
    int p = iAlignUp(w, 128);
    size += h*p;
    sizeTmp += nd*h*p; 
  }
  float *memoryTmp = NULL; 
  size_t pitch;
  size += sizeTmp;
  safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size+4095)/4096*sizeof(float)));
  float *memorySub = memoryTmp + sizeTmp;

  CudaImage lowImg;
  lowImg.Allocate(width, height, iAlignUp(width, 128), false, memorySub);
  if (!scaleUp) {
    LowPass(lowImg, img, max(initBlur, 0.001f));
    ExtractSiftLoop(siftData, lowImg, numOctaves, 0.0f, thresh, lowestScale, 1.0f, memoryTmp, memorySub + height*iAlignUp(width, 128));
    safeCall(hipMemcpyFromSymbol(&siftData.numPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
    siftData.numPts = (siftData.numPts<siftData.maxPts ? siftData.numPts : siftData.maxPts);
  } else {
    CudaImage upImg;
    upImg.Allocate(width, height, iAlignUp(width, 128), false, memoryTmp);
    ScaleUp(upImg, img);
    LowPass(lowImg, upImg, max(initBlur, 0.001f));
    ExtractSiftLoop(siftData, lowImg, numOctaves, 0.0f, thresh, lowestScale*2.0f, 1.0f, memoryTmp, memorySub + height*iAlignUp(width, 128));
    safeCall(hipMemcpyFromSymbol(&siftData.numPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
    siftData.numPts = (siftData.numPts<siftData.maxPts ? siftData.numPts : siftData.maxPts);
    RescalePositions(siftData, 0.5f);
  }
  
  safeCall(hipFree(memoryTmp));
#ifdef MANAGEDMEM
  safeCall(hipDeviceSynchronize());
#else
  if (siftData.h_data)
    safeCall(hipMemcpy(siftData.h_data, siftData.d_data, sizeof(SiftPoint)*siftData.numPts, hipMemcpyDeviceToHost));
#endif
  double totTime = timer.read();
#ifndef VERBOSE
  printf("Total time incl memory =      %.2f ms\n", totTime);
#endif
}

extern double DynamicMain(CudaImage &img, SiftData &siftData, int numOctaves, double initBlur, float thresh, float lowestScale, float edgeLimit, float *memoryTmp);

void ExtractSiftLoop(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp, float *memorySub) 
{
  TimerGPU timer(0);
#if 1
  int w = img.width;
  int h = img.height;
  if (numOctaves>1) {
    CudaImage subImg;
    int p = iAlignUp(w/2, 128);
    subImg.Allocate(w/2, h/2, p, false, memorySub); 
    ScaleDown(subImg, img, 0.5f);
    float totInitBlur = (float)sqrt(initBlur*initBlur + 0.5f*0.5f) / 2.0f;
    ExtractSiftLoop(siftData, subImg, numOctaves-1, totInitBlur, thresh, lowestScale, subsampling*2.0f, memoryTmp, memorySub + (h/2)*p);
  }
  if (lowestScale<subsampling*2.0f) 
    ExtractSiftOctave(siftData, img, initBlur, thresh, lowestScale, subsampling, memoryTmp);
#else
  DynamicMain(img, siftData, numOctaves, initBlur, thresh, lowestScale, 10.0f, memoryTmp);
#endif
  double totTime = timer.read();
#ifdef VERBOSE
  printf("ExtractSift time total =      %.2f ms\n\n", totTime);
#endif
}

void ExtractSiftOctave(SiftData &siftData, CudaImage &img, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp)
{
  const int nd = NUM_SCALES + 3;
  TimerGPU timer0;
  CudaImage diffImg[nd];
  int w = img.width; 
  int h = img.height;
  int p = iAlignUp(w, 128);
  for (int i=0;i<nd-1;i++) 
    diffImg[i].Allocate(w, h, p, false, memoryTmp + i*p*h); 

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = img.d_data;
  resDesc.res.pitch2D.width = img.width;
  resDesc.res.pitch2D.height = img.height;
  resDesc.res.pitch2D.pitchInBytes = img.pitch*sizeof(float);  
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0]   = hipAddressModeClamp;
  texDesc.addressMode[1]   = hipAddressModeClamp;
  texDesc.filterMode       = hipFilterModeLinear;
  texDesc.readMode         = hipReadModeElementType;
  texDesc.normalizedCoords = 0;
  // Create texture object
  hipTextureObject_t texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  TimerGPU timer1;
  float baseBlur = pow(2.0f, -1.0f/NUM_SCALES);
  float diffScale = pow(2.0f, 1.0f/NUM_SCALES);
  LaplaceMulti(texObj, img, diffImg, baseBlur, diffScale, initBlur);
  int fstPts = 0;
  safeCall(hipMemcpyFromSymbol(&fstPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  double sigma = baseBlur*diffScale;
  FindPointsMulti(diffImg, siftData, thresh, 10.0f, sigma, 1.0f/NUM_SCALES, lowestScale/subsampling, subsampling);
  double gpuTimeDoG = timer1.read();
  TimerGPU timer4;
  int totPts = 0;
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
  if (totPts>fstPts) {
    ComputeOrientations(texObj, siftData, fstPts, totPts); 
    safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
    totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
    ExtractSiftDescriptors(texObj, siftData, fstPts, totPts, subsampling); 
  }
  safeCall(hipDestroyTextureObject(texObj));
  double gpuTimeSift = timer4.read();

  double totTime = timer0.read();
#ifdef VERBOSE
  printf("GPU time : %.2f ms + %.2f ms + %.2f ms = %.2f ms\n", totTime-gpuTimeDoG-gpuTimeSift, gpuTimeDoG, gpuTimeSift, totTime);
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
  if (totPts>0) 
    printf("           %.2f ms / DoG,  %.4f ms / Sift,  #Sift = %d\n", gpuTimeDoG/NUM_SCALES, gpuTimeSift/(totPts-fstPts), totPts-fstPts); 
#endif
}

void InitSiftData(SiftData &data, int num, bool host, bool dev)
{
  data.numPts = 0;
  data.maxPts = num;
  int sz = sizeof(SiftPoint)*num;
#ifdef MANAGEDMEM
  safeCall(hipMallocManaged((void **)&data.m_data, sz));
#else
  data.h_data = NULL;
  if (host)
    data.h_data = (SiftPoint *)malloc(sz);
  data.d_data = NULL;
  if (dev)
    safeCall(hipMalloc((void **)&data.d_data, sz));
#endif
}

void FreeSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
  safeCall(hipFree(data.m_data));
#else
  if (data.d_data!=NULL)
    safeCall(hipFree(data.d_data));
  data.d_data = NULL;
  if (data.h_data!=NULL)
    free(data.h_data);
#endif
  data.numPts = 0;
  data.maxPts = 0;
}

void PrintSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
  SiftPoint *h_data = data.m_data;
#else
  SiftPoint *h_data = data.h_data;
  if (data.h_data==NULL) {
    h_data = (SiftPoint *)malloc(sizeof(SiftPoint)*data.maxPts);
    safeCall(hipMemcpy(h_data, data.d_data, sizeof(SiftPoint)*data.numPts, hipMemcpyDeviceToHost));
    data.h_data = h_data;
  }
#endif
  for (int i=0;i<data.numPts;i++) {
    printf("xpos         = %.2f\n", h_data[i].xpos);
    printf("ypos         = %.2f\n", h_data[i].ypos);
    printf("scale        = %.2f\n", h_data[i].scale);
    printf("sharpness    = %.2f\n", h_data[i].sharpness);
    printf("edgeness     = %.2f\n", h_data[i].edgeness);
    printf("orientation  = %.2f\n", h_data[i].orientation);
    printf("score        = %.2f\n", h_data[i].score);
    float *siftData = (float*)&h_data[i].data;
    for (int j=0;j<8;j++) {
      if (j==0) 
	printf("data = ");
      else 
	printf("       ");
      for (int k=0;k<16;k++)
	if (siftData[j+8*k]<0.05)
	  printf(" .   ");
	else
	  printf("%.2f ", siftData[j+8*k]);
      printf("\n");
    }
  }
  printf("Number of available points: %d\n", data.numPts);
  printf("Number of allocated points: %d\n", data.maxPts);
}

///////////////////////////////////////////////////////////////////////////////
// Host side master functions
///////////////////////////////////////////////////////////////////////////////

double ScaleDown(CudaImage &res, CudaImage &src, float variance)
{
  if (res.d_data==NULL || src.d_data==NULL) {
    printf("ScaleDown: missing data\n");
    return 0.0;
  }
  float h_Kernel[5];
  float kernelSum = 0.0f;
  for (int j=0;j<5;j++) {
    h_Kernel[j] = (float)expf(-(double)(j-2)*(j-2)/2.0/variance);      
    kernelSum += h_Kernel[j];
  }
  for (int j=0;j<5;j++)
    h_Kernel[j] /= kernelSum;  
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel1), h_Kernel, 5*sizeof(float)));
  dim3 blocks(iDivUp(src.width, SCALEDOWN_W), iDivUp(src.height, SCALEDOWN_H));
  dim3 threads(SCALEDOWN_W + 4);
  ScaleDown<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch); 
  checkMsg("ScaleDown() execution failed\n");
  return 0.0;
}

double ScaleUp(CudaImage &res, CudaImage &src)
{
  if (res.d_data==NULL || src.d_data==NULL) {
    printf("ScaleUp: missing data\n");
    return 0.0;
  }
  dim3 blocks(iDivUp(res.width, SCALEUP_W), iDivUp(res.height, SCALEUP_H));
  dim3 threads(SCALEUP_W, SCALEUP_H);
  ScaleUp<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch); 
  checkMsg("ScaleUp() execution failed\n");
  return 0.0;
}


double ComputeOrientations(hipTextureObject_t texObj, SiftData &siftData, int fstPts, int totPts)
{
  dim3 blocks(totPts - fstPts);
  dim3 threads(128);
#ifdef MANAGEDMEM
  ComputeOrientations<<<blocks, threads>>>(texObj, siftData.m_data, fstPts);
#else
  ComputeOrientations<<<blocks, threads>>>(texObj, siftData.d_data, fstPts);
#endif
  checkMsg("ComputeOrientations() execution failed\n");
  return 0.0;
}

double ExtractSiftDescriptors(hipTextureObject_t texObj, SiftData &siftData, int fstPts, int totPts, float subsampling)
{
  dim3 blocks(totPts - fstPts); 
  dim3 threads(16, 8);
#ifdef MANAGEDMEM
  ExtractSiftDescriptors<<<blocks, threads>>>(texObj, siftData.m_data, fstPts, subsampling);
#else
  ExtractSiftDescriptors<<<blocks, threads>>>(texObj, siftData.d_data, fstPts, subsampling);
#endif
  checkMsg("ExtractSiftDescriptors() execution failed\n");
  return 0.0; 
}

double RescalePositions(SiftData &siftData, float scale)
{
  dim3 blocks(iDivUp(siftData.numPts, 64));
  dim3 threads(64);
  RescalePositions<<<blocks, threads>>>(siftData.d_data, siftData.numPts, scale);
  checkMsg("RescapePositions() execution failed\n");
  return 0.0; 
}

double LowPass(CudaImage &res, CudaImage &src, float scale)
{
  float kernel[16];
  float kernelSum = 0.0f;
  float ivar2 = 1.0f/(2.0f*scale*scale);
  for (int j=-LOWPASS_R;j<=LOWPASS_R;j++) {
    kernel[j+LOWPASS_R] = (float)expf(-(double)j*j*ivar2);
    kernelSum += kernel[j+LOWPASS_R]; 
  }
  for (int j=-LOWPASS_R;j<=LOWPASS_R;j++) 
    kernel[j+LOWPASS_R] /= kernelSum;  
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel2), kernel, 12*16*sizeof(float)));
  int width = res.width;
  int pitch = res.pitch;
  int height = res.height;
  dim3 blocks(iDivUp(width, LOWPASS_W), iDivUp(height, LOWPASS_H));
  dim3 threads(LOWPASS_W+2*LOWPASS_R, LOWPASS_H);
  LowPass<<<blocks, threads>>>(src.d_data, res.d_data, width, pitch, height);
  checkMsg("LowPass() execution failed\n");
  return 0.0; 
}

//==================== Multi-scale functions ===================//

double LaplaceMulti(hipTextureObject_t texObj, CudaImage &baseImage, CudaImage *results, float baseBlur, float diffScale, float initBlur)
{
  float kernel[12*16];
  float scale = baseBlur;
  for (int i=0;i<NUM_SCALES+3;i++) {
    float kernelSum = 0.0f;
    float var = scale*scale - initBlur*initBlur;
    for (int j=-LAPLACE_R;j<=LAPLACE_R;j++) {
      kernel[16*i+j+LAPLACE_R] = (float)expf(-(double)j*j/2.0/var);
      kernelSum += kernel[16*i+j+LAPLACE_R]; 
    }
    for (int j=-LAPLACE_R;j<=LAPLACE_R;j++) 
      kernel[16*i+j+LAPLACE_R] /= kernelSum;  
    scale *= diffScale;
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel2), kernel, 12*16*sizeof(float)));
  int width = results[0].width;
  int pitch = results[0].pitch;
  int height = results[0].height;
  dim3 blocks(iDivUp(width, LAPLACE_W), height);
  dim3 threads(LAPLACE_W+2*LAPLACE_R, LAPLACE_S);
#if 1
  LaplaceMultiMem<<<blocks, threads>>>(baseImage.d_data, results[0].d_data, width, pitch, height);
#else
  LaplaceMultiTex<<<blocks, threads>>>(texObj, results[0].d_data, width, pitch, height);
#endif
  checkMsg("LaplaceMulti() execution failed\n");
  return 0.0; 
}

double FindPointsMulti(CudaImage *sources, SiftData &siftData, float thresh, float edgeLimit, float scale, float factor, float lowestScale, float subsampling)
{
  if (sources->d_data==NULL) {
    printf("FindPointsMulti: missing data\n");
    return 0.0;
  }
  int w = sources->width;
  int p = sources->pitch;
  int h = sources->height;
  float threshs[2] = { thresh, -thresh };
  float scales[NUM_SCALES];  
  float diffScale = pow(2.0f, factor);
  for (int i=0;i<NUM_SCALES;i++) {
    scales[i] = scale;
    scale *= diffScale;
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Threshold), &threshs, 2*sizeof(float)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_EdgeLimit), &edgeLimit, sizeof(float)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Scales), scales, sizeof(float)*NUM_SCALES));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Factor), &factor, sizeof(float)));

  dim3 blocks(iDivUp(w, MINMAX_W)*NUM_SCALES, iDivUp(h, MINMAX_H));
  dim3 threads(MINMAX_W + 2); 
#ifdef MANAGEDMEM
  FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.m_data, w, p, h, NUM_SCALES, subsampling, lowestScale); 
#else
  FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.d_data, w, p, h, NUM_SCALES, subsampling, lowestScale); 
#endif
  checkMsg("FindPointsMulti() execution failed\n");
  return 0.0;
}

