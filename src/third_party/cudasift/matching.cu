#include "hip/hip_runtime.h"
#include "cudaSift.h"
#include "cudautils.h"

//================= Device matching functions =====================//

__global__ void MatchSiftPoints(SiftPoint *sift1, SiftPoint *sift2, float *corrData, int numPts1, int numPts2)
{
  __shared__ float siftPoint[128];
  __shared__ float sums[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int p1 = blockIdx.x;
  const int p2 = blockIdx.y*16 + ty;
  const float *ptr1 = sift1[p1].data;
  const float *ptr2 = sift2[p2].data;
  const int i = 16*ty + tx;
  if (ty<8)
    siftPoint[i] = ptr1[i];
  __syncthreads();
  float sum = 0.0f;
  if (p2<numPts2)
    for (int j=0;j<8;j++)
      sum += siftPoint[16*j+tx] * ptr2[16*j+tx];
  sums[i] = sum;
  __syncthreads();
  if (tx<8)
    sums[i] += sums[i+8];
  __syncthreads();
  if (tx<4)
    sums[i] += sums[i+4];
  __syncthreads();
  if (ty==0) {
    sum = sums[16*tx+0] + sums[16*tx+1] + sums[16*tx+2] + sums[16*tx+3];
    corrData[p1*gridDim.y*16 + blockIdx.y*16 + tx] = sum;
  }
  __syncthreads();
}


__global__ void MatchSiftPoints2(SiftPoint *sift1, SiftPoint *sift2, float *corrData, int numPts1, int numPts2)
{
  __shared__ float siftPoints1[16*128];
  __shared__ float siftPoints2[16*128];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const float *ptr1 = sift1[min(numPts1-1,blockIdx.x*16 + ty)].data;
  const float *ptr2 = sift2[min(numPts2-1,blockIdx.y*16 + ty)].data;
  for (int i=0;i<8;i++) {
    siftPoints1[128*ty+16*i+tx] = ptr1[16*i+tx];
    siftPoints2[128*ty+16*i+tx] = ptr2[16*i+tx];
  }
  __syncthreads();
  const int p1 = blockIdx.x*16 + ty;
  const int p2 = blockIdx.y*16 + tx;
  const float *pt1 = &siftPoints1[ty*128];
  const float *pt2 = &siftPoints2[tx*128];
  float sum = 0.0f;
  for (int i=0;i<128;i++) {
    int itx = (i + tx)&127; // avoid bank conflicts
    sum += pt1[itx]*pt2[itx];
  }
  if (p1<numPts1)
    corrData[p1*gridDim.y*16 + p2] = (p2<numPts2 ? sum : -1.0f);
}

__global__ void FindMaxCorr(float *corrData, SiftPoint *sift1, SiftPoint *sift2, int numPts1, int corrWidth, int siftSize)
{
  __shared__ float maxScore[16*16];
  __shared__ float maxScor2[16*16];
  __shared__ int maxIndex[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int idx = ty*16 + tx;
  int p1 = blockIdx.x*16 + threadIdx.y;
  p1 = (p1>=numPts1 ? numPts1-1 : p1);
  maxScore[idx] = -1.0f;
  maxScor2[idx] = -1.0f;
  maxIndex[idx] = -1;
  __syncthreads();
  float *corrs = &corrData[p1*corrWidth];
  for (int i=tx;i<corrWidth;i+=16) {
    float val = corrs[i];
    if (val>maxScore[idx]) {
      maxScor2[idx] = maxScore[idx];
      maxScore[idx] = val;
      maxIndex[idx] = i;
    } else if (val>maxScor2[idx])
      maxScor2[idx] = val;
  }
  //if (p1==1)
  //  printf("tx = %d, score = %.2f, scor2 = %.2f, index = %d\n", 
  //	   tx, maxScore[idx], maxScor2[idx], maxIndex[idx]);
  __syncthreads();
  for (int len=8;len>0;len/=2) {
    if (tx<8) {
      float val = maxScore[idx+len];
      int i = maxIndex[idx+len];
      if (val>maxScore[idx]) {
	maxScor2[idx] = maxScore[idx];
	maxScore[idx] = val;
	maxIndex[idx] = i;
      } else if (val>maxScor2[idx])
	maxScor2[idx] = val;
      float va2 = maxScor2[idx+len];
      if (va2>maxScor2[idx])
	maxScor2[idx] = va2;
    }
    __syncthreads();
    //if (p1==1 && tx<len) 
    //  printf("tx = %d, score = %.2f, scor2 = %.2f, index = %d\n", 
    //	     tx, maxScore[idx], maxScor2[idx], maxIndex[idx]);
  }
  if (tx==6)
    sift1[p1].score = maxScore[ty*16];
  if (tx==7)
    sift1[p1].ambiguity = maxScor2[ty*16] / (maxScore[ty*16] + 1e-6);
  if (tx==8)
    sift1[p1].match = maxIndex[ty*16];
  if (tx==9)
    sift1[p1].match_xpos = sift2[maxIndex[ty*16]].xpos;
  if (tx==10)
    sift1[p1].match_ypos = sift2[maxIndex[ty*16]].ypos;
  __syncthreads();
  //if (tx==0)
  //  printf("index = %d/%d, score = %.2f, ambiguity = %.2f, match = %d\n", 
  //	p1, numPts1, sift1[p1].score, sift1[p1].ambiguity, sift1[p1].match);
}

template <int size>
__device__ void InvertMatrix(float elem[size][size], float res[size][size]) 
{  
  int indx[size];
  float b[size];
  float vv[size];
  for (int i=0;i<size;i++)
    indx[i] = 0;
  int imax = 0;
  float d = 1.0;
  for (int i=0;i<size;i++) { // find biggest element for each row
    float big = 0.0;
    for (int j=0;j<size;j++) {
      float temp = fabs(elem[i][j]); 
      if (temp>big) 
	big = temp;
    }
    if (big>0.0)
      vv[i] = 1.0/big;
    else
      vv[i] = 1e16;
  }
  for (int j=0;j<size;j++) { 
    for (int i=0;i<j;i++) { // i<j
      float sum = elem[i][j]; // i<j (lower left)
      for (int k=0;k<i;k++) // k<i<j
	sum -= elem[i][k]*elem[k][j]; // i>k (upper right), k<j (lower left)
      elem[i][j] = sum; // i<j (lower left)
    }
    float big = 0.0;
    for (int i=j;i<size;i++) { // i>=j
      float sum = elem[i][j]; // i>=j (upper right)
      for (int k=0;k<j;k++) // k<j<=i
	sum -= elem[i][k]*elem[k][j]; // i>k (upper right), k<j (lower left)
      elem[i][j] = sum; // i>=j (upper right)
      float dum = vv[i]*fabs(sum);
      if (dum>=big) {
	big = dum;
	imax = i;  
      }
    }
    if (j!=imax) { // imax>j
      for (int k=0;k<size;k++) {
	float dum = elem[imax][k]; // upper right and lower left
	elem[imax][k] = elem[j][k];
	elem[j][k] = dum;
      }
      d = -d;
      vv[imax] = vv[j];
    }
    indx[j] = imax;
    if (elem[j][j]==0.0)  // j==j (upper right)
      elem[j][j] = 1e-16;
    if (j!=(size-1)) {
      float dum = 1.0/elem[j][j];
      for (int i=j+1;i<size;i++) // i>j
	elem[i][j] *= dum; // i>j (upper right)
    }
  }
  for (int j=0;j<size;j++) {
    for (int k=0;k<size;k++) 
      b[k] = 0.0;  
    b[j] = 1.0;
    int ii = -1;
    for (int i=0;i<size;i++) {
      int ip = indx[i];
      float sum = b[ip];
      b[ip] = b[i];
      if (ii!=-1)
	for (int j=ii;j<i;j++) 
	  sum -= elem[i][j]*b[j]; // i>j (upper right)
      else if (sum!=0.0)
        ii = i;
      b[i] = sum;
    }
    for (int i=size-1;i>=0;i--) {
      float sum = b[i];
      for (int j=i+1;j<size;j++) 
	sum -= elem[i][j]*b[j]; // i<j (lower left)
      b[i] = sum/elem[i][i]; // i==i (upper right)
    }
    for (int i=0;i<size;i++)
      res[i][j] = b[i];
  }
}

__global__ void ComputeHomographies(float *coord, int *randPts, float *homo, 
  int numPts) 
{
  float a[8][8], ia[8][8];
  float b[8]; 
  const int bx = blockIdx.x;
  const int tx = threadIdx.x;
  const int idx = blockDim.x*bx + tx;
  const int numLoops = blockDim.x*gridDim.x;
  for (int i=0;i<4;i++) {
    int pt = randPts[i*numLoops+idx];
    float x1 = coord[pt+0*numPts];
    float y1 = coord[pt+1*numPts];
    float x2 = coord[pt+2*numPts];
    float y2 = coord[pt+3*numPts];
    float *row1 = a[2*i+0];
    row1[0] = x1;
    row1[1] = y1;
    row1[2] = 1.0;
    row1[3] = row1[4] = row1[5] = 0.0;
    row1[6] = -x2*x1;
    row1[7] = -x2*y1;
    float *row2 = a[2*i+1];
    row2[0] = row2[1] = row2[2] = 0.0;
    row2[3] = x1;
    row2[4] = y1;
    row2[5] = 1.0;
    row2[6] = -y2*x1;
    row2[7] = -y2*y1;
    b[2*i+0] = x2;
    b[2*i+1] = y2;
  }
  InvertMatrix<8>(a, ia);
  __syncthreads();
  for (int j=0;j<8;j++) {
    float sum = 0.0f;
    for (int i=0;i<8;i++) 
      sum += ia[j][i]*b[i];
    homo[j*numLoops+idx] = sum;
  }
  __syncthreads();
}

#define TESTHOMO_TESTS 16 // number of tests per block,  alt. 32, 32
#define TESTHOMO_LOOPS 16 // number of loops per block,  alt.  8, 16 

__global__ void TestHomographies(float *d_coord, float *d_homo, 
  int *d_counts, int numPts, float thresh2)
{
  __shared__ float homo[8*TESTHOMO_LOOPS];
  __shared__ int cnts[TESTHOMO_TESTS*TESTHOMO_LOOPS];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int idx = blockIdx.y*blockDim.y + tx;
  const int numLoops = blockDim.y*gridDim.y;
  if (ty<8 && tx<TESTHOMO_LOOPS)
    homo[tx*8+ty] = d_homo[idx+ty*numLoops];
  __syncthreads();
  float a[8];
  for (int i=0;i<8;i++) 
    a[i] = homo[ty*8+i];
  int cnt = 0;
  for (int i=tx;i<numPts;i+=TESTHOMO_TESTS) {
    float x1 = d_coord[i+0*numPts];
    float y1 = d_coord[i+1*numPts];
    float x2 = d_coord[i+2*numPts];
    float y2 = d_coord[i+3*numPts];
    float nomx = __fmul_rz(a[0],x1) + __fmul_rz(a[1],y1) + a[2];
    float nomy = __fmul_rz(a[3],x1) + __fmul_rz(a[4],y1) + a[5];
    float deno = __fmul_rz(a[6],x1) + __fmul_rz(a[7],y1) + 1.0f;
    float errx = __fmul_rz(x2,deno) - nomx;
    float erry = __fmul_rz(y2,deno) - nomy;
    float err2 = __fmul_rz(errx,errx) + __fmul_rz(erry,erry);
    if (err2<__fmul_rz(thresh2,__fmul_rz(deno,deno)))
      cnt ++;
  }
  int kty = TESTHOMO_TESTS*ty;
  cnts[kty + tx] = cnt;
  __syncthreads();
  int len = TESTHOMO_TESTS/2;
  while (len>0) {
    if (tx<len)
      cnts[kty + tx] += cnts[kty + tx + len];
    len /= 2;
    __syncthreads();
  }
  if (tx<TESTHOMO_LOOPS && ty==0)
    d_counts[idx] = cnts[TESTHOMO_TESTS*tx];
  __syncthreads();
}

//================= Host matching functions =====================//

double FindHomography(SiftData &data, float *homography, int *numMatches, int numLoops, float minScore, float maxAmbiguity, float thresh)
{
  *numMatches = 0;
  homography[0] = homography[4] = homography[8] = 1.0f;
  homography[1] = homography[2] = homography[3] = 0.0f;
  homography[5] = homography[6] = homography[7] = 0.0f;
#ifdef MANAGEDMEM
  SiftPoint *d_sift = data.m_data;
#else
  if (data.d_data==NULL)
    return 0.0f;
  SiftPoint *d_sift = data.d_data;
#endif
  TimerGPU timer(0);
  numLoops = iDivUp(numLoops,16)*16;
  int numPts = data.numPts;
  if (numPts<8)
    return 0.0f;
  int numPtsUp = iDivUp(numPts, 16)*16;
  float *d_coord, *d_homo;
  int *d_randPts, *h_randPts;
  int randSize = 4*sizeof(int)*numLoops;
  int szFl = sizeof(float);
  int szPt = sizeof(SiftPoint);
  safeCall(hipMalloc((void **)&d_coord, 4*sizeof(float)*numPtsUp));
  safeCall(hipMalloc((void **)&d_randPts, randSize));
  safeCall(hipMalloc((void **)&d_homo, 8*sizeof(float)*numLoops));
  h_randPts = (int*)malloc(randSize);
  float *h_scores = (float *)malloc(sizeof(float)*numPtsUp);
  float *h_ambiguities = (float *)malloc(sizeof(float)*numPtsUp);
  safeCall(hipMemcpy2D(h_scores, szFl, &d_sift[0].score, szPt, szFl, numPts, hipMemcpyDeviceToHost));
  safeCall(hipMemcpy2D(h_ambiguities, szFl, &d_sift[0].ambiguity, szPt, szFl, numPts, hipMemcpyDeviceToHost));
  int *validPts = (int *)malloc(sizeof(int)*numPts);
  int numValid = 0;
  for (int i=0;i<numPts;i++) {
    if (h_scores[i]>minScore && h_ambiguities[i]<maxAmbiguity)
      validPts[numValid++] = i;
  }
  free(h_scores);
  free(h_ambiguities);
  if (numValid>=8) {
    for (int i=0;i<numLoops;i++) {
      int p1 = rand() % numValid;
      int p2 = rand() % numValid;
      int p3 = rand() % numValid;
      int p4 = rand() % numValid;
      while (p2==p1) p2 = rand() % numValid;
      while (p3==p1 || p3==p2) p3 = rand() % numValid;
      while (p4==p1 || p4==p2 || p4==p3) p4 = rand() % numValid;
      h_randPts[i+0*numLoops] = validPts[p1];
      h_randPts[i+1*numLoops] = validPts[p2];
      h_randPts[i+2*numLoops] = validPts[p3];
      h_randPts[i+3*numLoops] = validPts[p4];
    }
    safeCall(hipMemcpy(d_randPts, h_randPts, randSize, hipMemcpyHostToDevice));
    safeCall(hipMemcpy2D(&d_coord[0*numPtsUp], szFl, &d_sift[0].xpos, szPt, szFl, numPts, hipMemcpyDeviceToDevice));
    safeCall(hipMemcpy2D(&d_coord[1*numPtsUp], szFl, &d_sift[0].ypos, szPt, szFl, numPts, hipMemcpyDeviceToDevice));
    safeCall(hipMemcpy2D(&d_coord[2*numPtsUp], szFl, &d_sift[0].match_xpos, szPt, szFl, numPts, hipMemcpyDeviceToDevice));
    safeCall(hipMemcpy2D(&d_coord[3*numPtsUp], szFl, &d_sift[0].match_ypos, szPt, szFl, numPts, hipMemcpyDeviceToDevice));
    ComputeHomographies<<<numLoops/16, 16>>>(d_coord, d_randPts, d_homo, numPtsUp);
    safeCall(hipDeviceSynchronize());
    checkMsg("ComputeHomographies() execution failed\n");
    dim3 blocks(1, numLoops/TESTHOMO_LOOPS);
    dim3 threads(TESTHOMO_TESTS, TESTHOMO_LOOPS);
    TestHomographies<<<blocks, threads>>>(d_coord, d_homo, d_randPts, numPtsUp, thresh*thresh);
    safeCall(hipDeviceSynchronize());
    checkMsg("TestHomographies() execution failed\n");
    safeCall(hipMemcpy(h_randPts, d_randPts, sizeof(int)*numLoops, hipMemcpyDeviceToHost));
    int maxIndex = -1, maxCount = -1;
    for (int i=0;i<numLoops;i++) 
      if (h_randPts[i]>maxCount) {
	maxCount = h_randPts[i];
	maxIndex = i;
      }
    *numMatches = maxCount;
    safeCall(hipMemcpy2D(homography, szFl, &d_homo[maxIndex], sizeof(float)*numLoops, szFl, 8, hipMemcpyDeviceToHost));
  }
  free(validPts);
  free(h_randPts);
  safeCall(hipFree(d_homo));
  safeCall(hipFree(d_randPts));
  safeCall(hipFree(d_coord));
  double gpuTime = timer.read();
#ifdef VERBOSE
  printf("FindHomography time =         %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

double MatchSiftData(SiftData &data1, SiftData &data2)
{
  TimerGPU timer(0);
  int numPts1 = data1.numPts;
  int numPts2 = data2.numPts;
  if (!numPts1 || !numPts2) 
    return 0.0;
#ifdef MANAGEDMEM
  SiftPoint *sift1 = data1.m_data;
  SiftPoint *sift2 = data2.m_data;
#else
  if (data1.d_data==NULL || data2.d_data==NULL)
    return 0.0f;
  SiftPoint *sift1 = data1.d_data;
  SiftPoint *sift2 = data2.d_data;
#endif
  
  float *d_corrData; 
  int corrWidth = iDivUp(numPts2, 16)*16;
  int corrSize = sizeof(float)*numPts1*corrWidth;
  safeCall(hipMalloc((void **)&d_corrData, corrSize));
#if 0
  dim3 blocks1(numPts1, iDivUp(numPts2, 16));
  dim3 threads1(16, 16); // each block: 1 points x 16 points
  MatchSiftPoints<<<blocks1, threads1>>>(sift1, sift2, d_corrData, numPts1, numPts2);
#else
  dim3 blocks(iDivUp(numPts1,16), iDivUp(numPts2, 16));
  dim3 threads(16, 16); // each block: 1 points x 16 points
  MatchSiftPoints2<<<blocks, threads>>>(sift1, sift2, d_corrData, numPts1, numPts2);
#endif
  safeCall(hipDeviceSynchronize());
  dim3 blocksMax(iDivUp(numPts1, 16));
  dim3 threadsMax(16, 16);
  FindMaxCorr<<<blocksMax, threadsMax>>>(d_corrData, sift1, sift2, numPts1, corrWidth, sizeof(SiftPoint));
  safeCall(hipDeviceSynchronize());
  checkMsg("MatchSiftPoints() execution failed\n");
  safeCall(hipFree(d_corrData));
  if (data1.h_data!=NULL) {
    float *h_ptr = &data1.h_data[0].score;
    float *d_ptr = &data1.d_data[0].score;
    safeCall(hipMemcpy2D(h_ptr, sizeof(SiftPoint), d_ptr, sizeof(SiftPoint), 5*sizeof(float), data1.numPts, hipMemcpyDeviceToHost));
  }

  double gpuTime = timer.read();
#ifdef VERBOSE
  printf("MatchSiftData time =          %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}		 
  
