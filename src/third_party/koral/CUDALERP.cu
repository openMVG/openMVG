#include "hip/hip_runtime.h"
/*******************************************************************
*   CUDALERP.cu
*   KORAL
*
*	Author: Kareem Omar
*	kareem.omar@uah.edu
*	https://github.com/komrad36
*
*	Last updated Dec 27, 2016
*******************************************************************/
//
// ## Summary ##
// KORAL is a novel, extremely fast, highly accurate, scale- and
// rotation-invariant, CPU-GPU cooperative detector-descriptor.
//
// Detection is based on the author's custom multi-scale KFAST corner
// detector, with rapid bilinear interpolation performed by the GPU
// asynchronously while the CPU works on KFAST.
//
// ## Usage ##
// Basic use of KORAL is extremely easy, although, of course, for a
// larger high-performance pipeline, users will benefit from
// calling KORAL functions directly and modifying it to suit their needs.
//
// To detect and describe, simply #include "KORAL.h" and
// then do:
//
// 	    KORAL koral(scale_factor, scale_levels);
//      koral.go(image, width, height, KFAST_threshold);
//
// where scale_factor is the factor by which each scale leve
// is reduced from the previous, scale_levels is the total
// number of such scale levels used, image is a pointer to
// uint8_t (grayscale) image data, and KFAST_threshold
// is the threshold supplied to the KFAST feature detector.
//
// After this call, keypoints are avaiable in a vector at 
// koral.kps, while descriptors are available at
// koral.desc.
//
// Portions of KORAL require SSE, AVX, AVX2, and CUDA.
// The author is working on reduced-performance versions
// with lesser requirements, but as the intent of this work
// is primarily novel performance capability, modern
// hardware and this full version are highly recommended.
//
// Description is performed by the GPU using the novel CLATCH
// (CUDA LATCH) binary descriptor kernel.
//
// Rotation invariance is provided by a novel vectorized
// SSE angle weight detector.
//
// All components have been written and carefully tuned by the author
// for maximum performance and have no external dependencies. Some have
// been modified for integration into KORAL,
// but the original standalone projects are all availble on
// the author's GitHub (https://github.com/komrad36).
//
// These individual components are:
// -KFAST        (https://github.com/komrad36/KFAST)
// -CUDALERP     (https://github.com/komrad36/CUDALERP)
// -FeatureAngle (https://github.com/komrad36/FeatureAngle)
// -CLATCH       (https://github.com/komrad36/CLATCH)
//
// In addition, the natural next step of matching descriptors
// is available in the author's currently separate
// project, CUDAK2NN (https://github.com/komrad36/CUDAK2NN).
//
// A key insight responsible for much of the performance of
// this insanely fast system is due to Christopher Parker
// (https://github.com/csp256), to whom I am extremely grateful.
// 
// The file 'main.cpp' is a simple test driver
// illustrating example usage.It requires OpenCV
// for image read and keypoint display.KORAL itself,
// however, does not require OpenCV or any other
// external dependencies.
//
// Note that KORAL is a work in progress.
// Suggestions and improvements are welcomed.
//
// ## License ##
// The FAST detector was created by Edward Rosten and Tom Drummond
// as described in the 2006 paper by Rosten and Drummond:
// "Machine learning for high-speed corner detection"
//         Edward Rosten and Tom Drummond
// https://www.edwardrosten.com/work/rosten_2006_machine.pdf
//
// The FAST detector is BSD licensed:
// 
// Copyright(c) 2006, 2008, 2009, 2010 Edward Rosten
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met :
// 
// 
// *Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
// 
// *Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and / or other materials provided with the distribution.
// 
// *Neither the name of the University of Cambridge nor the names of
// its contributors may be used to endorse or promote products derived
// from this software without specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
// A PARTICULAR PURPOSE ARE DISCLAIMED.IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES(INCLUDING, BUT NOT LIMITED TO,
// 	PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// 	PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// 	LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT(INCLUDING
// 		NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// 	SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
//
//
//
// KORAL is licensed under the MIT License : https://opensource.org/licenses/mit-license.php
// 
// Copyright(c) 2016 Kareem Omar, Christopher Parker
// 
// Permission is hereby granted, free of charge,
// to any person obtaining a copy of this software and associated documentation
// files(the "Software"), to deal in the Software without restriction, including
// without limitation the rights to use, copy, modify, merge, publish, distribute,
// sublicense, and / or sell copies of the Software, and to permit persons to whom
// the Software is furnished to do so, subject to the following conditions :
// 
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
// INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A
// PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
// OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
// SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
//
// 
// Note again that KORAL is a work in progress.
// Suggestions and improvements are welcomed.
//

#include "CUDALERP.h"

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
CUDALERP_kernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, uint8_t* __restrict const d_out, const size_t pitch, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;
	const float fy = (y + 0.5f)*gys - 0.5f;
	const float wt_y = fy - floor(fy);
	const float invwt_y = 1.0f - wt_y;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fx = (x + 0.5f)*gxs - 0.5f;
		// less accurate and not really much (or any) faster
		// -----------------
		// const float res = tex2D<float>(d_img_tex, fx, fy);
		// -----------------
		const float4 f = tex2Dgather<float4>(d_img_tex, fx + 0.5f, fy + 0.5f);
		const float wt_x = fx - floor(fx);
		const float invwt_x = 1.0f - wt_x;
		const float xa = invwt_x*f.w + wt_x*f.z;
		const float xb = invwt_x*f.x + wt_x*f.y;
		const float res = 255.0f*(invwt_y*xa + wt_y*xb) + 0.5f;
		// -----------------
		if (x < neww) d_out[y*pitch + x] = res;
	}
}

void CUDALERP(const hipTextureObject_t d_img_tex, const float gxs, const float gys, uint8_t* __restrict const d_out, const size_t pitch, const uint32_t neww, const uint32_t newh, const hipStream_t stream) {
	CUDALERP_kernel<<<{((neww - 1) >> 9) + 1, newh}, 256, 0, stream>>>(d_img_tex, gxs, gys, d_out, pitch, neww);
}
