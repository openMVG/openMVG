#include "hip/hip_runtime.h"
/*******************************************************************
*   CLATCH.cu
*   KORAL
*
*	Author: Kareem Omar
*	kareem.omar@uah.edu
*	https://github.com/komrad36
*
*	Last updated Dec 27, 2016
*******************************************************************/
//
// ## Summary ##
// KORAL is a novel, extremely fast, highly accurate, scale- and
// rotation-invariant, CPU-GPU cooperative detector-descriptor.
//
// Detection is based on the author's custom multi-scale KFAST corner
// detector, with rapid bilinear interpolation performed by the GPU
// asynchronously while the CPU works on KFAST.
//
// ## Usage ##
// Basic use of KORAL is extremely easy, although, of course, for a
// larger high-performance pipeline, users will benefit from
// calling KORAL functions directly and modifying it to suit their needs.
//
// To detect and describe, simply #include "KORAL.h" and
// then do:
//
// 	    KORAL koral(scale_factor, scale_levels);
//      koral.go(image, width, height, KFAST_threshold);
//
// where scale_factor is the factor by which each scale leve
// is reduced from the previous, scale_levels is the total
// number of such scale levels used, image is a pointer to
// uint8_t (grayscale) image data, and KFAST_threshold
// is the threshold supplied to the KFAST feature detector.
//
// After this call, keypoints are avaiable in a vector at 
// koral.kps, while descriptors are available at
// koral.desc.
//
// Portions of KORAL require SSE, AVX, AVX2, and CUDA.
// The author is working on reduced-performance versions
// with lesser requirements, but as the intent of this work
// is primarily novel performance capability, modern
// hardware and this full version are highly recommended.
//
// Description is performed by the GPU using the novel CLATCH
// (CUDA LATCH) binary descriptor kernel.
//
// Rotation invariance is provided by a novel vectorized
// SSE angle weight detector.
//
// All components have been written and carefully tuned by the author
// for maximum performance and have no external dependencies. Some have
// been modified for integration into KORAL,
// but the original standalone projects are all availble on
// the author's GitHub (https://github.com/komrad36).
//
// These individual components are:
// -KFAST        (https://github.com/komrad36/KFAST)
// -CUDALERP     (https://github.com/komrad36/CUDALERP)
// -FeatureAngle (https://github.com/komrad36/FeatureAngle)
// -CLATCH       (https://github.com/komrad36/CLATCH)
//
// In addition, the natural next step of matching descriptors
// is available in the author's currently separate
// project, CUDAK2NN (https://github.com/komrad36/CUDAK2NN).
//
// A key insight responsible for much of the performance of
// this insanely fast system is due to Christopher Parker
// (https://github.com/csp256), to whom I am extremely grateful.
// 
// The file 'main.cpp' is a simple test driver
// illustrating example usage.It requires OpenCV
// for image read and keypoint display.KORAL itself,
// however, does not require OpenCV or any other
// external dependencies.
//
// Note that KORAL is a work in progress.
// Suggestions and improvements are welcomed.
//
// ## License ##
// The FAST detector was created by Edward Rosten and Tom Drummond
// as described in the 2006 paper by Rosten and Drummond:
// "Machine learning for high-speed corner detection"
//         Edward Rosten and Tom Drummond
// https://www.edwardrosten.com/work/rosten_2006_machine.pdf
//
// The FAST detector is BSD licensed:
// 
// Copyright(c) 2006, 2008, 2009, 2010 Edward Rosten
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met :
// 
// 
// *Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
// 
// *Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and / or other materials provided with the distribution.
// 
// *Neither the name of the University of Cambridge nor the names of
// its contributors may be used to endorse or promote products derived
// from this software without specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
// A PARTICULAR PURPOSE ARE DISCLAIMED.IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES(INCLUDING, BUT NOT LIMITED TO,
// 	PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// 	PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// 	LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT(INCLUDING
// 		NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// 	SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
//
//
//
// KORAL is licensed under the MIT License : https://opensource.org/licenses/mit-license.php
// 
// Copyright(c) 2016 Kareem Omar, Christopher Parker
// 
// Permission is hereby granted, free of charge,
// to any person obtaining a copy of this software and associated documentation
// files(the "Software"), to deal in the Software without restriction, including
// without limitation the rights to use, copy, modify, merge, publish, distribute,
// sublicense, and / or sell copies of the Software, and to permit persons to whom
// the Software is furnished to do so, subject to the following conditions :
// 
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
// INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A
// PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
// OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
// SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
//
// 
// Note again that KORAL is a work in progress.
// Suggestions and improvements are welcomed.
//

#include "CLATCH.h"

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(512, 4)
#endif
CLATCH_kernel(const hipTextureObject_t d_all_tex[8], const hipTextureObject_t d_triplets, const Keypoint* const __restrict d_kps, uint32_t* const __restrict__ d_desc) {
	volatile __shared__ uint8_t s_ROI[4608];
	const Keypoint pt = d_kps[blockIdx.x];
	const hipTextureObject_t d_img_tex = d_all_tex[pt.scale];
	const float s = sin(pt.angle), c = cos(pt.angle);
	for (int32_t i = 0; i <= 48; i += 16) {
		for (int32_t k = 0; k <= 32; k += 32) {
			const float x_offset = static_cast<float>(static_cast<int>(threadIdx.x) + k - 32);
			const float y_offset = static_cast<float>(static_cast<int>(threadIdx.y) + i - 32);
			s_ROI[(threadIdx.y + i) * 72 + threadIdx.x + k] = tex2D<uint8_t>(d_img_tex, static_cast<int>((pt.x + (x_offset*c - y_offset*s)) + 0.5f), static_cast<int>((pt.y + (x_offset*s + y_offset*c)) + 0.5f));
		}
	}
	uint32_t ROI_base = 144 * (threadIdx.x & 3) + (threadIdx.x >> 2), triplet_base = threadIdx.y << 5, desc = 0;
	__syncthreads();
	for (int32_t i = 0; i < 4; ++i, triplet_base += 8) {
		int32_t accum[8];
		for (uint32_t j = 0; j < 8; ++j) {
			const ushort4 t = tex1D<ushort4>(d_triplets, triplet_base + j);
			const int32_t b1 = s_ROI[ROI_base + t.y],      b2 = s_ROI[ROI_base + t.y + 72]     ;
			const int32_t a1 = s_ROI[ROI_base + t.x] - b1, a2 = s_ROI[ROI_base + t.x + 72] - b2;
			const int32_t c1 = s_ROI[ROI_base + t.z] - b1, c2 = s_ROI[ROI_base + t.z + 72] - b2;
			accum[j] = a1 * a1 - c1 * c1 + a2 * a2 - c2 * c2;
		}
		for (int32_t k = 1; k <= 4; k <<= 1) {
			for (int32_t s = 0; s < 8; s += k) accum[s] += __shfl_xor(accum[s], k);
			if (threadIdx.x & k) for (int32_t s = 0; s < 8; s += k << 1) accum[s] = accum[s + k];
		}
		accum[0] += __shfl_xor(accum[0], 8);
		desc |= (accum[0] + __shfl_xor(accum[0], 16) < 0) << ((i << 3) + (threadIdx.x & 7));
	}
	for (int32_t s = 1; s <= 4; s <<= 1) desc |= __shfl_xor(desc, s);
	if (threadIdx.x == 0) d_desc[(blockIdx.x << 4) + threadIdx.y] = desc;
}

void CLATCH(hipTextureObject_t d_all_tex[8], const hipTextureObject_t d_triplets, const Keypoint* const __restrict d_kps, const int num_kps, uint64_t* const __restrict d_desc) {
	CLATCH_kernel<<<num_kps, { 32, 16 } >>>(d_all_tex, d_triplets, d_kps, reinterpret_cast<uint32_t*>(d_desc));
}
